#include "hip/hip_runtime.h"
/*
 *  Copyright (C) 2017 Jesus Martin Berlanga. All Rights Reserved.
 *
 *  This file is part of Paracrypt.
 *
 *  Paracrypt is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  Paracrypt is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with Paracrypt.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

#include "CudaAes8B.cuh"
#include "utils/cuda_logging.cuh"

__global__ void __cuda_aes_8b_ptr_encrypt__(
		const paracrypt::BlockCipher::Mode m,
		unsigned int n,
		uint32_t offset,
		const uint32_t* d,
		uint32_t* out,
		uint32_t* neigh,
		uint32_t* iv,
		uint32_t* k,
		const int key_bits,
		uint32_t* T0,
		uint32_t* T1,
		uint32_t* T2,
		uint32_t* T3
    )
{
	// Each block has its own shared memory
	// We have an state for each two threads
	extern __shared__ uint32_t state[];

	unsigned int bi = ((blockIdx.x * blockDim.x) + threadIdx.x); // section index

	const int s_size = blockDim.x/2;
	//__LOG_TRACE__("s_size => %d", s_size);
	uint32_t* s0 = state           ;
	uint32_t* s1 = state+(  s_size);
	uint32_t* s2 = state+(2*s_size);
	uint32_t* s3 = state+(3*s_size);
	uint32_t* t0 = state+(4*s_size);
	uint32_t* t1 = state+(5*s_size);
	uint32_t* t2 = state+(6*s_size);
	uint32_t* t3 = state+(7*s_size);

	unsigned int p = bi*2;
	uint32_t sti = threadIdx.x/2; //state index
	int ti = threadIdx.x%2; // block-thread index: 0 or 1 (2 threads per cipher-block)
	int valid_thread = bi < n*2;
	unsigned int key_index_sum = 0;

	uint8_t* s0p = (uint8_t*) &s0[sti];
	uint8_t* s1p = (uint8_t*) &s1[sti];
	uint8_t* s2p = (uint8_t*) &s2[sti];
	uint8_t* s3p = (uint8_t*) &s3[sti];
	uint8_t* t0p = (uint8_t*) &t0[sti];
	uint8_t* t1p = (uint8_t*) &t1[sti];
	uint8_t* t2p = (uint8_t*) &t2[sti];
	uint8_t* t3p = (uint8_t*) &t3[sti];

#if defined(DEBUG) && defined(DEVEL)
	if(valid_thread) {
    	__LOG_TRACE__("p %d: threadIx.x => %d",p,threadIdx.x);
    	__LOG_TRACE__("p %d: ti => %d",p,ti);
    }
#endif

	/*
	 * map byte array block to cipher state
	 * and add initial round key:
	 */
	if(m == paracrypt::BlockCipher::CTR || m == paracrypt::BlockCipher::GCM) {
		// - The counter is the block index: offset+bi
		// - For security the counter is combined with a noence: global_bi ^ iv
        // - Initial round: ^k
		uint32_t global_bi = offset+sti;
		if(valid_thread && ti == 0) {
			__LOG_TRACE__("p %d: global_bi => 0x%04x",p,global_bi);
			__LOG_TRACE__("p %d: iv[0] => 0x%04x",p,iv[0]);
			__LOG_TRACE__("p %d: iv[1] => 0x%04x",p,iv[1]);
			__LOG_TRACE__("p %d: k[0] => 0x%04x",p,k[0]);
			__LOG_TRACE__("p %d: k[1] => 0x%04x",p,k[1]);
			s0[sti] = global_bi ^ iv[0] ^ k[0];
			s1[sti] = global_bi ^ iv[1] ^ k[1];
		}
		else if(valid_thread && ti == 1) {
			__LOG_TRACE__("p %d: global_bi => 0x%04x",p,global_bi);
			__LOG_TRACE__("p %d: iv[2] => 0x%04x",p,iv[2]);
			__LOG_TRACE__("p %d: iv[3] => 0x%04x",p,iv[3]);
			__LOG_TRACE__("p %d: k[2] => 0x%04x",p,k[2]);
			__LOG_TRACE__("p %d: k[3] => 0x%04x",p,k[3]);
			s2[sti] = global_bi ^ iv[2] ^ k[2];
			s3[sti] = global_bi ^ iv[3] ^ k[3];
		}
	}
	else if(m == paracrypt::BlockCipher::CFB) {
		if(valid_thread && ti == 0) {
			if(bi == 0) {
				__LOG_TRACE__("p %d: iv[0] => 0x%04x",p,iv[0]);
				__LOG_TRACE__("p %d: iv[1] => 0x%04x",p,iv[1]);
				s0[sti] = iv[0] ^ k[0];
				s1[sti] = iv[1] ^ k[1];
			}
			else {
				if(sti == 0) { // && d == out) { TODO support for out-of-place
					// previous cipher-block is in another
					//  thread-block so we cannot __syncthreads()
					// and we use this data to ensure the data
					// we access is not overwritten
					int np = (blockIdx.x*4)-4;
					s0[sti] = neigh[np  ] ^ k[0];
					s1[sti] = neigh[np+1] ^ k[1];
					__LOG_TRACE__("p %d (bi %d): accessing neighbor at np %d.",p,bi,np);
				}
				else {
					s0[sti] = d[p-4] ^ k[0];
					s1[sti] = d[p-3] ^ k[1];
					__LOG_TRACE__("p %d (bi %d): accessing prev. block.",p,bi);
				}
			}
		}
		else if(valid_thread && ti == 1) {
			if(bi == 1) {
				__LOG_TRACE__("p %d: iv[2] => 0x%04x",p,iv[2]);
				__LOG_TRACE__("p %d: iv[3] => 0x%04x",p,iv[3]);
				s2[sti] = iv[2] ^ k[2];
				s3[sti] = iv[3] ^ k[3];
			}
			else {
				if(sti == 0) {
					int np = (blockIdx.x*4)-4;
					s2[sti] = neigh[np+2] ^ k[2];
					s3[sti] = neigh[np+3] ^ k[3];
					__LOG_TRACE__("p %d (bi %d): accessing neighbor at np %d.",p,bi,np);
				}
				else {
					s2[sti] = d[p-4] ^ k[2];
					s3[sti] = d[p-3] ^ k[3];
					__LOG_TRACE__("p %d (bi %d): accessing prev. block.",p,bi);
				}
			}
		}
	}
	else {
		if(valid_thread && ti == 0) {
			__LOG_TRACE__("p %d: ti => %d",p,ti);
			__LOG_TRACE__("p %d: d[0] => 0x%04x",p,d[p]);
			__LOG_TRACE__("p %d: d[1] => 0x%04x",p,d[p+1]);
			__LOG_TRACE__("p %d: k[0] => 0x%04x",p,k[0]);
			__LOG_TRACE__("p %d: k[1] => 0x%04x",p,k[1]);
			s0[sti] = d[p]   ^ k[0];
			s1[sti] = d[p+1] ^ k[1];
			__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0[sti]);
			__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1[sti]);
		}
		else if(valid_thread && ti == 1) {
			__LOG_TRACE__("p %d: d[2] => 0x%04x",p,d[p]);
			__LOG_TRACE__("p %d: d[3] => 0x%04x",p,d[p+1]);
			__LOG_TRACE__("p %d: k[2] => 0x%04x",p,k[2]);
			__LOG_TRACE__("p %d: k[3] => 0x%04x",p,k[3]);
			s2[sti] = d[p]   ^ k[2];
			s3[sti] = d[p+1] ^ k[3];
			__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2[sti]);
			__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3[sti]);
		}
	}

	// 8 rounds - in each loop we do two rounds
	#pragma unroll
	for(int r2 = 1; r2 <= 4; r2++) {
		__syncthreads();
		if(valid_thread && ti == 0) {
			__LOG_TRACE__("p %d: (s0      ) & 0xff => 0x%04x",p,s0p[0]);
			__LOG_TRACE__("p %d: (s1 >>  8) & 0xff => 0x%04x",p,s1p[1]);
			__LOG_TRACE__("p %d: (s2 >> 16) & 0xff => 0x%04x",p,s2p[2]);
			__LOG_TRACE__("p %d: (s3 >> 24)        => 0x%04x",p,(s3[sti] >> 24));
			__LOG_TRACE__("p %d: T0[(s0      ) & 0xff] => 0x%04x",p,T0[s0p[0]]);
			__LOG_TRACE__("p %d: T1[(s1 >>  8) & 0xff] => 0x%04x",p,T1[s1p[1]]);
			__LOG_TRACE__("p %d: T2[(s2 >> 16) & 0xff] => 0x%04x",p,T2[s2p[2]]);
			__LOG_TRACE__("p %d: T3[(s3 >> 24)       ] => 0x%04x",p,T3[s3p[3]]);
			__LOG_TRACE__("p %d: k[%d] => 0x%04x",p,(r2*8)-4 , k[(r2*8)-4]);
			t0[sti] =
				T0[s0p[0]] ^
				T1[s1p[1]] ^
				T2[s2p[2]] ^
				T3[s3p[3]] ^
				k[(r2*8)-4];
			t1[sti] =
				T0[s1p[0]] ^
				T1[s2p[1]] ^
				T2[s3p[2]] ^
				T3[s0p[3]] ^
				k[(r2*8)-3];
			__LOG_TRACE__("p %d: t0 => 0x%04x",p,t0[sti]);
			__LOG_TRACE__("p %d: t1 => 0x%04x",p,t1[sti]);
		}
		else if(valid_thread && ti == 1) {
		__LOG_TRACE__("p %d: (s0      ) & 0xff => 0x%04x",p,s0p[0]);
		__LOG_TRACE__("p %d: (s1 >>  8) & 0xff => 0x%04x",p,s1p[1]);
		__LOG_TRACE__("p %d: (s2 >> 16) & 0xff => 0x%04x",p,s2p[2]);
		__LOG_TRACE__("p %d: (s3 >> 24)        => 0x%04x",p,(s3[sti] >> 24));
		__LOG_TRACE__("p %d: T0[(s0      ) & 0xff] => 0x%04x",p,T0[s0p[0]]);
		__LOG_TRACE__("p %d: T1[(s1 >>  8) & 0xff] => 0x%04x",p,T1[s1p[1]]);
		__LOG_TRACE__("p %d: T2[(s2 >> 16) & 0xff] => 0x%04x",p,T2[s2p[2]]);
		__LOG_TRACE__("p %d: T3[(s3 >> 24)       ] => 0x%04x",p,T3[s3p[3]]);
		__LOG_TRACE__("p %d: k[%d] => 0x%04x",p,(r2*8)-4 , k[(r2*8)-4]);
		t2[sti] =
			T0[s2p[0]] ^
			T1[s3p[1]] ^
			T2[s0p[2]] ^
			T3[s1p[3]] ^
			k[(r2*8)-2];
		t3[sti] =
			T0[s3p[0]] ^
			T1[s0p[1]] ^
			T2[s1p[2]] ^
			T3[s2p[3]] ^
			k[(r2*8)-1];
		__LOG_TRACE__("p %d: t2 => 0x%04x",p,t2[sti]);
		__LOG_TRACE__("p %d: t3 => 0x%04x",p,t3[sti]);
		}
		__syncthreads();
		if(valid_thread && ti == 0) {
			s0[sti] =
				T0[t0p[0]] ^
				T1[t1p[1]] ^
				T2[t2p[2]] ^
				T3[t3p[3]] ^
				k[(r2*8)  ];
			s1[sti] =
				T0[t1p[0]] ^
				T1[t2p[1]] ^
				T2[t3p[2]] ^
				T3[t0p[3]] ^
				k[(r2*8)+1];
			__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0[sti]);
			__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1[sti]);
		}
		else if(valid_thread && ti == 1) {
			s2[sti] =
				T0[t2p[0]] ^
				T1[t3p[1]] ^
				T2[t0p[2]] ^
				T3[t1p[3]] ^
				k[(r2*8)+2];
			s3[sti] =
				T0[t3p[0]] ^
				T1[t0p[1]] ^
				T2[t1p[2]] ^
				T3[t2p[3]] ^
				k[(r2*8)+3];
			__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2[sti]);
			__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3[sti]);
		}
	}

	if(key_bits >= 192) {
		key_index_sum = 8;
		__syncthreads();
		if(valid_thread && ti == 0) {
			__LOG_TRACE__("p %d: (s0      ) & 0xff => 0x%04x",p,s0p[0]);
			__LOG_TRACE__("p %d: (s1 >>  8) & 0xff => 0x%04x",p,s1p[1]);
			__LOG_TRACE__("p %d: (s2 >> 16) & 0xff => 0x%04x",p,s2p[2]);
			__LOG_TRACE__("p %d: (s3 >> 24)        => 0x%04x",p,(s3[sti] >> 24));
			__LOG_TRACE__("p %d: T0[(s0      ) & 0xff] => 0x%04x",p,T0[s0p[0]]);
			__LOG_TRACE__("p %d: T1[(s1 >>  8) & 0xff] => 0x%04x",p,T1[s1p[1]]);
			__LOG_TRACE__("p %d: T2[(s2 >> 16) & 0xff] => 0x%04x",p,T2[s2p[2]]);
			__LOG_TRACE__("p %d: T3[(s3 >> 24)       ] => 0x%04x",p,T3[s3p[3]]);
			__LOG_TRACE__("p %d: k[36] => 0x%04x",p, k[36]);
			t0[sti] =
				T0[s0p[0]] ^
				T1[s1p[1]] ^
				T2[s2p[2]] ^
				T3[s3p[3]] ^
				k[36];
			t1[sti] =
				T0[s1p[0]] ^
				T1[s2p[1]] ^
				T2[s3p[2]] ^
				T3[s0p[3]] ^
				k[37];
			__LOG_TRACE__("p %d: t0 => 0x%04x",p,t0[sti]);
			__LOG_TRACE__("p %d: t1 => 0x%04x",p,t1[sti]);
		}
		else if(valid_thread && ti == 1) {
		__LOG_TRACE__("p %d: (s0      ) & 0xff => 0x%04x",p,s0p[0]);
		__LOG_TRACE__("p %d: (s1 >>  8) & 0xff => 0x%04x",p,s1p[1]);
		__LOG_TRACE__("p %d: (s2 >> 16) & 0xff => 0x%04x",p,s2p[2]);
		__LOG_TRACE__("p %d: (s3 >> 24)        => 0x%04x",p,(s3[sti] >> 24));
		__LOG_TRACE__("p %d: T0[(s0      ) & 0xff] => 0x%04x",p,T0[s0p[0]]);
		__LOG_TRACE__("p %d: T1[(s1 >>  8) & 0xff] => 0x%04x",p,T1[s1p[1]]);
		__LOG_TRACE__("p %d: T2[(s2 >> 16) & 0xff] => 0x%04x",p,T2[s2p[2]]);
		__LOG_TRACE__("p %d: T3[(s3 >> 24)       ] => 0x%04x",p,T3[s3p[3]]);
		__LOG_TRACE__("p %d: k[38] => 0x%04x",p, k[38]);
		t2[sti] =
			T0[s2p[0]] ^
			T1[s3p[1]] ^
			T2[s0p[2]] ^
			T3[s1p[3]] ^
			k[38];
		t3[sti] =
			T0[s3p[0]] ^
			T1[s0p[1]] ^
			T2[s1p[2]] ^
			T3[s2p[3]] ^
			k[39];
		__LOG_TRACE__("p %d: t2 => 0x%04x",p,t2[sti]);
		__LOG_TRACE__("p %d: t3 => 0x%04x",p,t3[sti]);
		}
		__syncthreads();
		if(valid_thread && ti == 0) {
			s0[sti] =
				T0[t0p[0]] ^
				T1[t1p[1]] ^
				T2[t2p[2]] ^
				T3[t3p[3]] ^
				k[40];
			s1[sti] =
				T0[t1p[0]] ^
				T1[t2p[1]] ^
				T2[t3p[2]] ^
				T3[t0p[3]] ^
				k[41];
			__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0[sti]);
			__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1[sti]);
		}
		else if(valid_thread && ti == 1) {
			s2[sti] =
				T0[t2p[0]] ^
				T1[t3p[1]] ^
				T2[t0p[2]] ^
				T3[t1p[3]] ^
				k[42];
			s3[sti] =
				T0[t3p[0]] ^
				T1[t0p[1]] ^
				T2[t1p[2]] ^
				T3[t2p[3]] ^
				k[43];
			__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2[sti]);
			__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3[sti]);
		}

		if(key_bits == 256) {
			key_index_sum = 16;
			__syncthreads();
			if(valid_thread && ti == 0) {
				__LOG_TRACE__("p %d: (s0      ) & 0xff => 0x%04x",p,s0p[0]);
				__LOG_TRACE__("p %d: (s1 >>  8) & 0xff => 0x%04x",p,s1p[1]);
				__LOG_TRACE__("p %d: (s2 >> 16) & 0xff => 0x%04x",p,s2p[2]);
				__LOG_TRACE__("p %d: (s3 >> 24)        => 0x%04x",p,(s3[sti] >> 24));
				__LOG_TRACE__("p %d: T0[(s0      ) & 0xff] => 0x%04x",p,T0[s0p[0]]);
				__LOG_TRACE__("p %d: T1[(s1 >>  8) & 0xff] => 0x%04x",p,T1[s1p[1]]);
				__LOG_TRACE__("p %d: T2[(s2 >> 16) & 0xff] => 0x%04x",p,T2[s2p[2]]);
				__LOG_TRACE__("p %d: T3[(s3 >> 24)       ] => 0x%04x",p,T3[s3p[3]]);
				__LOG_TRACE__("p %d: k[44] => 0x%04x",p, k[44]);
				t0[sti] =
					T0[s0p[0]] ^
					T1[s1p[1]] ^
					T2[s2p[2]] ^
					T3[s3p[3]] ^
					k[44];
				t1[sti] =
					T0[s1p[0]] ^
					T1[s2p[1]] ^
					T2[s3p[2]] ^
					T3[s0p[3]] ^
					k[45];
				__LOG_TRACE__("p %d: t0 => 0x%04x",p,t0[sti]);
				__LOG_TRACE__("p %d: t1 => 0x%04x",p,t1[sti]);
			}
			else if(valid_thread && ti == 1) {
			__LOG_TRACE__("p %d: (s0      ) & 0xff => 0x%04x",p,s0p[0]);
			__LOG_TRACE__("p %d: (s1 >>  8) & 0xff => 0x%04x",p,s1p[1]);
			__LOG_TRACE__("p %d: (s2 >> 16) & 0xff => 0x%04x",p,s2p[2]);
			__LOG_TRACE__("p %d: (s3 >> 24)        => 0x%04x",p,(s3[sti] >> 24));
			__LOG_TRACE__("p %d: T0[(s0      ) & 0xff] => 0x%04x",p,T0[s0p[0]]);
			__LOG_TRACE__("p %d: T1[(s1 >>  8) & 0xff] => 0x%04x",p,T1[s1p[1]]);
			__LOG_TRACE__("p %d: T2[(s2 >> 16) & 0xff] => 0x%04x",p,T2[s2p[2]]);
			__LOG_TRACE__("p %d: T3[(s3 >> 24)       ] => 0x%04x",p,T3[s3p[3]]);
			__LOG_TRACE__("p %d: k[46] => 0x%04x",p, k[46]);
			t2[sti] =
				T0[s2p[0]] ^
				T1[s3p[1]] ^
				T2[s0p[2]] ^
				T3[s1p[3]] ^
				k[46];
			t3[sti] =
				T0[s3p[0]] ^
				T1[s0p[1]] ^
				T2[s1p[2]] ^
				T3[s2p[3]] ^
				k[47];
			__LOG_TRACE__("p %d: t2 => 0x%04x",p,t2[sti]);
			__LOG_TRACE__("p %d: t3 => 0x%04x",p,t3[sti]);
			}
			__syncthreads();
			if(valid_thread && ti == 0) {
				s0[sti] =
					T0[t0p[0]] ^
					T1[t1p[1]] ^
					T2[t2p[2]] ^
					T3[t3p[3]] ^
					k[48];
				s1[sti] =
					T0[t1p[0]] ^
					T1[t2p[1]] ^
					T2[t3p[2]] ^
					T3[t0p[3]] ^
					k[49];
				__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0[sti]);
				__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1[sti]);
			}
			else if(valid_thread && ti == 1) {
				s2[sti] =
					T0[t2p[0]] ^
					T1[t3p[1]] ^
					T2[t0p[2]] ^
					T3[t1p[3]] ^
					k[50];
				s3[sti] =
					T0[t3p[0]] ^
					T1[t0p[1]] ^
					T2[t1p[2]] ^
					T3[t2p[3]] ^
					k[51];
				__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2[sti]);
				__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3[sti]);
			}
		}
	}

	__syncthreads();
	if(valid_thread && ti == 0) {
		t0[sti] =
			T0[s0p[0]] ^
			T1[s1p[1]] ^
			T2[s2p[2]] ^
			T3[s3p[3]] ^
			k[36+key_index_sum];
		t1[sti] =
			T0[s1p[0]] ^
			T1[s2p[1]] ^
			T2[s3p[2]] ^
			T3[s0p[3]] ^
			k[37+key_index_sum];
		__LOG_TRACE__("p %d: t0 => 0x%04x",p,t0[sti]);
		__LOG_TRACE__("p %d: t1 => 0x%04x",p,t1[sti]);
	}
	if(valid_thread && ti == 1) {
		t2[sti] =
			T0[s2p[0]] ^
			T1[s3p[1]] ^
			T2[s0p[2]] ^
			T3[s1p[3]] ^
			k[38+key_index_sum];
		t3[sti] =
			T0[s3p[0]] ^
			T1[s0p[1]] ^
			T2[s1p[2]] ^
			T3[s2p[3]] ^
			k[39+key_index_sum];
		__LOG_TRACE__("p %d: t2 => 0x%04x",p,t2[sti]);
		__LOG_TRACE__("p %d: t3 => 0x%04x",p,t3[sti]);
	}

	// last round - save result
	__syncthreads();
	if(valid_thread && ti == 0) {
		s0[sti] =
			(T2[t0p[0]] & 0x000000ff) ^
			(T3[t1p[1]] & 0x0000ff00) ^
			(T0[t2p[2]] & 0x00ff0000) ^
			(T1[t3p[3]] & 0xff000000) ^
			k[40+key_index_sum];
		__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0[sti]);
		s1[sti] =
			(T2[t1p[0]] & 0x000000ff) ^
			(T3[t2p[1]] & 0x0000ff00) ^
			(T0[t3p[2]] & 0x00ff0000) ^
			(T1[t0p[3]] & 0xff000000) ^
			k[41+key_index_sum];
		__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1);
	}
	else if(valid_thread && ti == 1){
		s2[sti] =
			(T2[t2p[0]] & 0x000000ff) ^
			(T3[t3p[1]] & 0x0000ff00) ^
			(T0[t0p[2]] & 0x00ff0000) ^
			(T1[t1p[3]] & 0xff000000) ^
			k[42+key_index_sum];
		__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2[sti]);
		s3[sti] =
			(T2[t3p[0]] & 0x000000ff) ^
			(T3[t0p[1]] & 0x0000ff00) ^
			(T0[t1p[2]] & 0x00ff0000) ^
			(T2[t2p[3]] & 0xff000000) ^
			k[43+key_index_sum];
		__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3[sti]);
	}

	if(		   m == paracrypt::BlockCipher::CTR
			|| m == paracrypt::BlockCipher::GCM
			|| m == paracrypt::BlockCipher::CFB
	){
		if(valid_thread && ti == 0) {
			__LOG_TRACE__("p %d: d[0] => 0x%04x",p,d[p  ]);
			__LOG_TRACE__("p %d: d[1] => 0x%04x",p,d[p+1]);
			s0[sti] ^= d[p  ];
			s1[sti] ^= d[p+1];
			__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0);
			__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1);
		}
		else if(valid_thread && ti == 1) {
			__LOG_TRACE__("p %d: d[0] => 0x%04x",p,d[p ]);
			__LOG_TRACE__("p %d: d[1] => 0x%04x",p,d[p+1]);
			s2[sti] ^= d[p  ];
			s3[sti] ^= d[p+1];
			__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2);
			__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3);
		}
	}

	if(valid_thread && ti == 0) {
		out[p] = s0[sti];
		out[p+1] = s1[sti];
	}
	else if(valid_thread && ti == 1) {
		out[p] = s2[sti];
		out[p+1] = s3[sti];
	}
}

__global__ void __cuda_aes_8b_ptr_decrypt__(
        const paracrypt::BlockCipher::Mode m,
		unsigned int n,
		unsigned int offset,
		const uint32_t* d,
		uint32_t* out,
		uint32_t* neigh,
		uint32_t* iv,
		uint32_t* k,
		const int key_bits,
		uint32_t* T0,
		uint32_t* T1,
		uint32_t* T2,
		uint32_t* T3,
		uint8_t* T4
    )
{
	// Each block has its own shared memory
	// We have an state for each two threads
	extern __shared__ uint32_t state[];

	unsigned int bi = ((blockIdx.x * blockDim.x) + threadIdx.x); // section index

	const int s_size = blockDim.x/2;
	//__LOG_TRACE__("s_size => %d", s_size);
	uint32_t* s0 = state           ;
	uint32_t* s1 = state+(  s_size);
	uint32_t* s2 = state+(2*s_size);
	uint32_t* s3 = state+(3*s_size);
	uint32_t* t0 = state+(4*s_size);
	uint32_t* t1 = state+(5*s_size);
	uint32_t* t2 = state+(6*s_size);
	uint32_t* t3 = state+(7*s_size);

	unsigned int p = bi*2;
	unsigned int sti = threadIdx.x/2; //state index
	int ti = threadIdx.x%2; // block-thread index: 0 or 1 (2 threads per cipher-block)
	int valid_thread = bi < n*2;
	unsigned int key_index_sum = 0;

	uint8_t* s0p = (uint8_t*) &s0[sti];
	uint8_t* s1p = (uint8_t*) &s1[sti];
	uint8_t* s2p = (uint8_t*) &s2[sti];
	uint8_t* s3p = (uint8_t*) &s3[sti];
	uint8_t* t0p = (uint8_t*) &t0[sti];
	uint8_t* t1p = (uint8_t*) &t1[sti];
	uint8_t* t2p = (uint8_t*) &t2[sti];
	uint8_t* t3p = (uint8_t*) &t3[sti];

#if defined(DEBUG) && defined(DEVEL)
	if(valid_thread) {
    	__LOG_TRACE__("p %d: threadIx.x => %d",p,threadIdx.x);
    	__LOG_TRACE__("p %d: ti => %d",p,ti);
    }
#endif

	/*
	 * map byte array block to cipher state
	 * and add initial round key:
	 */
	if(valid_thread && ti == 0) {
	__LOG_TRACE__("p %d: ti => %d",p,ti);
	__LOG_TRACE__("p %d: d[0] => 0x%04x",p,d[p]);
	__LOG_TRACE__("p %d: d[1] => 0x%04x",p,d[p+1]);
	__LOG_TRACE__("p %d: k[0] => 0x%04x",p,k[0]);
	__LOG_TRACE__("p %d: k[1] => 0x%04x",p,k[1]);
	s0[sti] = d[p]   ^ k[0];
	s1[sti] = d[p+1] ^ k[1];
	__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0[sti]);
	__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1[sti]);
	}
	else if(valid_thread && ti == 1) {
	__LOG_TRACE__("p %d: d[2] => 0x%04x",p,d[p]);
	__LOG_TRACE__("p %d: d[3] => 0x%04x",p,d[p+1]);
	__LOG_TRACE__("p %d: k[2] => 0x%04x",p,k[2]);
	__LOG_TRACE__("p %d: k[3] => 0x%04x",p,k[3]);
	s2[sti] = d[p]   ^ k[2];
	s3[sti] = d[p+1] ^ k[3];
	__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2[sti]);
	__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3[sti]);
	}

	// 8 rounds - in each loop we do two rounds
	#pragma unroll
	for(int r2 = 1; r2 <= 4; r2++) {
		__syncthreads();
		if(valid_thread && ti == 0) {
			__LOG_TRACE__("p %d: (s0      ) & 0xff => 0x%04x",p,s0p[0]);
			__LOG_TRACE__("p %d: (s1 >>  8) & 0xff => 0x%04x",p,s1p[1]);
			__LOG_TRACE__("p %d: (s2 >> 16) & 0xff => 0x%04x",p,s2p[2]);
			__LOG_TRACE__("p %d: (s3 >> 24)        => 0x%04x",p,(s3[sti] >> 24));
			__LOG_TRACE__("p %d: T0[(s0      ) & 0xff] => 0x%04x",p,T0[s0p[0]]);
			__LOG_TRACE__("p %d: T1[(s1 >>  8) & 0xff] => 0x%04x",p,T1[s1p[1]]);
			__LOG_TRACE__("p %d: T2[(s2 >> 16) & 0xff] => 0x%04x",p,T2[s2p[2]]);
			__LOG_TRACE__("p %d: T3[(s3 >> 24)       ] => 0x%04x",p,T3[s3p[3]]);
			__LOG_TRACE__("p %d: k[%d] => 0x%04x",p,(r2*8)-4 , k[(r2*8)-4]);
			t0[sti] =
				T0[s0p[0]] ^
				T1[s3p[1]] ^
				T2[s2p[2]] ^
				T3[s1p[3]] ^
				k[(r2*8)-4];
			t1[sti] =
				T0[s1p[0]] ^
				T1[s0p[1]] ^
				T2[s3p[2]] ^
				T3[s2p[3]] ^
				k[(r2*8)-3];
			__LOG_TRACE__("p %d: t0 => 0x%04x",p,t0[sti]);
			__LOG_TRACE__("p %d: t1 => 0x%04x",p,t1[sti]);
		}
		else if(valid_thread && ti == 1) {
		__LOG_TRACE__("p %d: (s0      ) & 0xff => 0x%04x",p,s0p[0]);
		__LOG_TRACE__("p %d: (s1 >>  8) & 0xff => 0x%04x",p,s1p[1]);
		__LOG_TRACE__("p %d: (s2 >> 16) & 0xff => 0x%04x",p,s2p[2]);
		__LOG_TRACE__("p %d: (s3 >> 24)        => 0x%04x",p,(s3[sti] >> 24));
		__LOG_TRACE__("p %d: T0[(s0      ) & 0xff] => 0x%04x",p,T0[s0p[0]]);
		__LOG_TRACE__("p %d: T1[(s1 >>  8) & 0xff] => 0x%04x",p,T1[s1p[1]]);
		__LOG_TRACE__("p %d: T2[(s2 >> 16) & 0xff] => 0x%04x",p,T2[s2p[2]]);
		__LOG_TRACE__("p %d: T3[(s3 >> 24)       ] => 0x%04x",p,T3[s3p[3]]);
		__LOG_TRACE__("p %d: k[%d] => 0x%04x",p,(r2*8)-4 , k[(r2*8)-4]);
		t2[sti] =
			T0[s2p[0]] ^
			T1[s1p[1]] ^
			T2[s0p[2]] ^
			T3[s3p[3]] ^
			k[(r2*8)-2];
		t3[sti] =
			T0[s3p[0]] ^
			T1[s2p[1]] ^
			T2[s1p[2]] ^
			T3[s0p[3]] ^
			k[(r2*8)-1];
		__LOG_TRACE__("p %d: t2 => 0x%04x",p,t2[sti]);
		__LOG_TRACE__("p %d: t3 => 0x%04x",p,t3[sti]);
		}
		__syncthreads();
		if(valid_thread && ti == 0) {
			s0[sti] =
				T0[t0p[0]] ^
				T1[t3p[1]] ^
				T2[t2p[2]] ^
				T3[t1p[3]] ^
				k[(r2*8)  ];
			s1[sti] =
				T0[t1p[0]] ^
				T1[t0p[1]] ^
				T2[t3p[2]] ^
				T3[t2p[3]] ^
				k[(r2*8)+1];
			__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0[sti]);
			__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1[sti]);
		}
		else if(valid_thread && ti == 1) {
			s2[sti] =
				T0[t2p[0]] ^
				T1[t1p[1]] ^
				T2[t0p[2]] ^
				T3[t3p[3]] ^
				k[(r2*8)+2];
			s3[sti] =
				T0[t3p[0]] ^
				T1[t2p[1]] ^
				T2[t1p[2]] ^
				T3[t0p[3]] ^
				k[(r2*8)+3];
			__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2[sti]);
			__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3[sti]);
		}
	}

	if(key_bits >= 192) {
		key_index_sum = 8;
		__syncthreads();
		if(valid_thread && ti == 0) {
			__LOG_TRACE__("p %d: (s0      ) & 0xff => 0x%04x",p,s0p[0]);
			__LOG_TRACE__("p %d: (s1 >>  8) & 0xff => 0x%04x",p,s1p[1]);
			__LOG_TRACE__("p %d: (s2 >> 16) & 0xff => 0x%04x",p,s2p[2]);
			__LOG_TRACE__("p %d: (s3 >> 24)        => 0x%04x",p,(s3[sti] >> 24));
			__LOG_TRACE__("p %d: T0[(s0      ) & 0xff] => 0x%04x",p,T0[s0p[0]]);
			__LOG_TRACE__("p %d: T1[(s1 >>  8) & 0xff] => 0x%04x",p,T1[s1p[1]]);
			__LOG_TRACE__("p %d: T2[(s2 >> 16) & 0xff] => 0x%04x",p,T2[s2p[2]]);
			__LOG_TRACE__("p %d: T3[(s3 >> 24)       ] => 0x%04x",p,T3[s3p[3]]);
			__LOG_TRACE__("p %d: k[36] => 0x%04x",p, k[36]);
			t0[sti] =
				T0[s0p[0]] ^
				T1[s3p[1]] ^
				T2[s2p[2]] ^
				T3[s1p[3]] ^
				k[36];
			t1[sti] =
				T0[s1p[0]] ^
				T1[s0p[1]] ^
				T2[s3p[2]] ^
				T3[s2p[3]] ^
				k[37];
			__LOG_TRACE__("p %d: t0 => 0x%04x",p,t0[sti]);
			__LOG_TRACE__("p %d: t1 => 0x%04x",p,t1[sti]);
		}
		else if(valid_thread && ti == 1) {
		__LOG_TRACE__("p %d: (s0      ) & 0xff => 0x%04x",p,s0p[0]);
		__LOG_TRACE__("p %d: (s1 >>  8) & 0xff => 0x%04x",p,s1p[1]);
		__LOG_TRACE__("p %d: (s2 >> 16) & 0xff => 0x%04x",p,s2p[2]);
		__LOG_TRACE__("p %d: (s3 >> 24)        => 0x%04x",p,(s3[sti] >> 24));
		__LOG_TRACE__("p %d: T0[(s0      ) & 0xff] => 0x%04x",p,T0[s0p[0]]);
		__LOG_TRACE__("p %d: T1[(s1 >>  8) & 0xff] => 0x%04x",p,T1[s1p[1]]);
		__LOG_TRACE__("p %d: T2[(s2 >> 16) & 0xff] => 0x%04x",p,T2[s2p[2]]);
		__LOG_TRACE__("p %d: T3[(s3 >> 24)       ] => 0x%04x",p,T3[s3p[3]]);
		__LOG_TRACE__("p %d: k[38] => 0x%04x",p, k[38]);
		t2[sti] =
			T0[s2p[0]] ^
			T1[s1p[1]] ^
			T2[s0p[2]] ^
			T3[s3p[3]] ^
			k[38];
		t3[sti] =
			T0[s3p[0]] ^
			T1[s2p[1]] ^
			T2[s1p[2]] ^
			T3[s0p[3]] ^
			k[39];
		__LOG_TRACE__("p %d: t2 => 0x%04x",p,t2[sti]);
		__LOG_TRACE__("p %d: t3 => 0x%04x",p,t3[sti]);
		}
		__syncthreads();
		if(valid_thread && ti == 0) {
			s0[sti] =
				T0[t0p[0]] ^
				T1[t3p[1]] ^
				T2[t2p[2]] ^
				T3[t1p[3]] ^
				k[40];
			s1[sti] =
				T0[t1p[0]] ^
				T1[t0p[1]] ^
				T2[t3p[2]] ^
				T3[t2p[3]] ^
				k[41];
			__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0[sti]);
			__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1[sti]);
		}
		else if(valid_thread && ti == 1) {
			s2[sti] =
				T0[t2p[0]] ^
				T1[t1p[1]] ^
				T2[t0p[2]] ^
				T3[t3p[3]] ^
				k[42];
			s3[sti] =
				T0[t3p[0]] ^
				T1[t2p[1]] ^
				T2[t1p[2]] ^
				T3[t0p[3]] ^
				k[43];
			__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2[sti]);
			__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3[sti]);
		}

		if(key_bits == 256) {
			key_index_sum = 16;
			__syncthreads();
			if(valid_thread && ti == 0) {
				__LOG_TRACE__("p %d: (s0      ) & 0xff => 0x%04x",p,s0p[0]);
				__LOG_TRACE__("p %d: (s1 >>  8) & 0xff => 0x%04x",p,s1p[1]);
				__LOG_TRACE__("p %d: (s2 >> 16) & 0xff => 0x%04x",p,s2p[2]);
				__LOG_TRACE__("p %d: (s3 >> 24)        => 0x%04x",p,(s3[sti] >> 24));
				__LOG_TRACE__("p %d: T0[(s0      ) & 0xff] => 0x%04x",p,T0[s0p[0]]);
				__LOG_TRACE__("p %d: T1[(s1 >>  8) & 0xff] => 0x%04x",p,T1[s1p[1]]);
				__LOG_TRACE__("p %d: T2[(s2 >> 16) & 0xff] => 0x%04x",p,T2[s2p[2]]);
				__LOG_TRACE__("p %d: T3[(s3 >> 24)       ] => 0x%04x",p,T3[s3p[3]]);
				__LOG_TRACE__("p %d: k[44] => 0x%04x",p, k[44]);
				t0[sti] =
					T0[s0p[0]] ^
					T1[s3p[1]] ^
					T2[s2p[2]] ^
					T3[s1p[3]] ^
					k[44];
				t1[sti] =
					T0[s1p[0]] ^
					T1[s0p[1]] ^
					T2[s3p[2]] ^
					T3[s2p[3]] ^
					k[45];
				__LOG_TRACE__("p %d: t0 => 0x%04x",p,t0[sti]);
				__LOG_TRACE__("p %d: t1 => 0x%04x",p,t1[sti]);
			}
			else if(valid_thread && ti == 1) {
			__LOG_TRACE__("p %d: (s0      ) & 0xff => 0x%04x",p,s0p[0]);
			__LOG_TRACE__("p %d: (s1 >>  8) & 0xff => 0x%04x",p,s1p[1]);
			__LOG_TRACE__("p %d: (s2 >> 16) & 0xff => 0x%04x",p,s2p[2]);
			__LOG_TRACE__("p %d: (s3 >> 24)        => 0x%04x",p,(s3[sti] >> 24));
			__LOG_TRACE__("p %d: T0[(s0      ) & 0xff] => 0x%04x",p,T0[s0p[0]]);
			__LOG_TRACE__("p %d: T1[(s1 >>  8) & 0xff] => 0x%04x",p,T1[s1p[1]]);
			__LOG_TRACE__("p %d: T2[(s2 >> 16) & 0xff] => 0x%04x",p,T2[s2p[2]]);
			__LOG_TRACE__("p %d: T3[(s3 >> 24)       ] => 0x%04x",p,T3[s3p[3]]);
			__LOG_TRACE__("p %d: k[46] => 0x%04x",p, k[46]);
			t2[sti] =
				T0[s2p[0]] ^
				T1[s1p[1]] ^
				T2[s0p[2]] ^
				T3[s3p[3]] ^
				k[46];
			t3[sti] =
				T0[s3p[0]] ^
				T1[s2p[1]] ^
				T2[s1p[2]] ^
				T3[s0p[3]] ^
				k[47];
			__LOG_TRACE__("p %d: t2 => 0x%04x",p,t2[sti]);
			__LOG_TRACE__("p %d: t3 => 0x%04x",p,t3[sti]);
			}
			__syncthreads();
			if(valid_thread && ti == 0) {
				s0[sti] =
					T0[t0p[0]] ^
					T1[t3p[1]] ^
					T2[t2p[2]] ^
					T3[t1p[3]] ^
					k[48];
				s1[sti] =
					T0[t1p[0]] ^
					T1[t0p[1]] ^
					T2[t3p[2]] ^
					T3[t2p[3]] ^
					k[49];
				__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0[sti]);
				__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1[sti]);
			}
			else if(valid_thread && ti == 1) {
				s2[sti] =
					T0[t2p[0]] ^
					T1[t1p[1]] ^
					T2[t0p[2]] ^
					T3[t3p[3]] ^
					k[50];
				s3[sti] =
					T0[t3p[0]] ^
					T1[t2p[1]] ^
					T2[t1p[2]] ^
					T3[t0p[3]] ^
					k[51];
				__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2[sti]);
				__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3[sti]);
			}
		}
	}

	__syncthreads();
	if(valid_thread && ti == 0) {
		t0[sti] =
			T0[s0p[0]] ^
			T1[s3p[1]] ^
			T2[s2p[2]] ^
			T3[s1p[3]] ^
			k[36+key_index_sum];
		t1[sti] =
			T0[s1p[0]] ^
			T1[s0p[1]] ^
			T2[s3p[2]] ^
			T3[s2p[3]] ^
			k[37+key_index_sum];
		__LOG_TRACE__("p %d: t0 => 0x%04x",p,t0[sti]);
		__LOG_TRACE__("p %d: t1 => 0x%04x",p,t1[sti]);
	}
	if(valid_thread && ti == 1) {
		t2[sti] =
			T0[s2p[0]] ^
			T1[s1p[1]] ^
			T2[s0p[2]] ^
			T3[s3p[3]] ^
			k[38+key_index_sum];
		t3[sti] =
			T0[s3p[0]] ^
			T1[s2p[1]] ^
			T2[s1p[2]] ^
			T3[s0p[3]] ^
			k[39+key_index_sum];
		__LOG_TRACE__("p %d: t2 => 0x%04x",p,t2[sti]);
		__LOG_TRACE__("p %d: t3 => 0x%04x",p,t3[sti]);
	}

	// last round - save result
	__syncthreads();
	if(valid_thread && ti == 0) {
		s0[sti] =
			((uint32_t)T4[t0p[0]]      ) ^
			((uint32_t)T4[t3p[1]] <<  8) ^
			((uint32_t)T4[t2p[2]] << 16) ^
			((uint32_t)T4[t1p[3]] << 24) ^
			k[40+key_index_sum];
		__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0[sti]);
		s1[sti] =
			((uint32_t)T4[t1p[0]]      ) ^
			((uint32_t)T4[t0p[1]] <<  8) ^
			((uint32_t)T4[t3p[2]] << 16) ^
			((uint32_t)T4[t2p[3]] << 24) ^
			k[41+key_index_sum];
		__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1);
	}
	else if(valid_thread && ti == 1){
		s2[sti] =
			((uint32_t)T4[t2p[0]]      ) ^
			((uint32_t)T4[t1p[1]] <<  8) ^
			((uint32_t)T4[t0p[2]] << 16) ^
			((uint32_t)T4[t3p[3]] << 24) ^
			k[42+key_index_sum];
		__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2[sti]);
		s3[sti] =
			((uint32_t)T4[t3p[0]]      ) ^
			((uint32_t)T4[t2p[1]] <<  8) ^
			((uint32_t)T4[t1p[2]] << 16) ^
			((uint32_t)T4[t0p[3]] << 24) ^
			k[43+key_index_sum];
		__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3[sti]);
	}

	/* if(m == paracrypt::BlockCipher::ECB) {
		// nothing in purpose
	}
	else */ if(m == paracrypt::BlockCipher::CBC) {
		if(valid_thread && ti == 0) {
			uint32_t c0,c1;
			if(bi == 0) {
				// there is no previous block - use input vector
				c0 = iv[0];
				c1 = iv[1];
				__LOG_TRACE__("p %d (sti %d): using IV.",p,sti);
			} else {
				// previous block
				if(sti == 0) { // && d == out) { TODO support for out-of-place
					// previous cipher-block is in another
					//  thread-block so we cannot __syncthreads()
					// and we use this data to ensure the data
					// we access is not overwritten
					int np = (blockIdx.x*4)-4;
					c0 = neigh[np  ];
					c1 = neigh[np+1];
					__LOG_TRACE__("p %d (sti %d): accessing neighbor at np %d.",p,sti,np);
				}
				else {
					c0 = d[p-4];
					c1 = d[p-3];
					__LOG_TRACE__("p %d (sti %d): accessing prev. block.",p,sti);
				}
			}
			__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0[sti]);
			__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1[sti]);
			__LOG_TRACE__("p %d: c0 => 0x%04x",p,c0);
			__LOG_TRACE__("p %d: c1 => 0x%04x",p,c1);
			s0[sti] ^= c0;
			s1[sti] ^= c1;
		}
		else if(valid_thread && ti == 1){
			uint32_t c2,c3;
			if(bi == 1) {
	            // there is no previous block - use input vector
				c2 = iv[2];
				c3 = iv[3];
				__LOG_TRACE__("p %d (sti %d): using IV.",p,sti);
			} else {
				// previous block
				if(sti == 0) { // && d == out) { TODO support for out-of-place
					// previous cipher-block is in another
					//  thread-block so we cannot __syncthreads()
					// and we use this data to ensure the data
					// we access is not overwritten
					int np = (blockIdx.x*4)-4;
					c2 = neigh[np+2];
					c3 = neigh[np+3];
					__LOG_TRACE__("p %d (sti %d): accessing neighbor at np %d.",p,sti,np);
				}
				else {
					c2 = d[p-4];
					c3 = d[p-3];
					__LOG_TRACE__("p %d (sti %d): accessing prev. block.",p,sti);
				}
			}
			__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2[sti]);
			__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3[sti]);
			__LOG_TRACE__("p %d: c2 => 0x%04x",p,c2);
			__LOG_TRACE__("p %d: c3 => 0x%04x",p,c3);
			s2[sti] ^= c2;
			s3[sti] ^= c3;
		}
		__syncthreads();
	}

	if(valid_thread && ti == 0) {
		out[p] = s0[sti];
		out[p+1] = s1[sti];
	}
	else if(valid_thread && ti == 1) {
		out[p] = s2[sti];
		out[p+1] = s3[sti];
	}
}

void cuda_aes_8b_ptr_encrypt(
		  paracrypt::BlockCipher::Mode m,
	  	  int gridSize,
	  	  int threadsPerBlock,
	  	  hipStream_t stream,
	  	  unsigned int n_blocks,
	  	  uint32_t offset,
	  	  unsigned char* in,
	  	  unsigned char* out,
	  	  unsigned char* neigh,
	  	  unsigned char* iv,
	  	  uint32_t* expanded_key,
	  	  int key_bits,
	  	  uint32_t* deviceTe0,
	  	  uint32_t* deviceTe1,
	  	  uint32_t* deviceTe2,
	  	  uint32_t* deviceTe3
	      )
{
	// (threadsPerBlock/2)4*2*sizeof(uint32_t);
	int shared_memory = threadsPerBlock*4*sizeof(uint32_t);
	__cuda_aes_8b_ptr_encrypt__<<<gridSize,threadsPerBlock,shared_memory,stream>>>(//*2>>>(
			m,
			n_blocks,
			offset,
			(const uint32_t*)in,
			(uint32_t*)out,
			(uint32_t*)neigh,
			(uint32_t*)iv,
			expanded_key,
			key_bits,
	   		deviceTe0,
	   		deviceTe1,
	   		deviceTe2,
	   		deviceTe3
	);
}

void cuda_aes_8b_ptr_decrypt(
		  paracrypt::BlockCipher::Mode m,
	  	  int gridSize,
	  	  int threadsPerBlock,
	  	  hipStream_t stream,
	  	  unsigned int n_blocks,
	  	  unsigned int offset,
	  	  unsigned char* in,
	  	  unsigned char* out,
	  	  unsigned char* neigh,
	  	  unsigned char* iv,
	  	  uint32_t* expanded_key,
	  	  int key_bits,
	  	  uint32_t* deviceTd0,
	  	  uint32_t* deviceTd1,
	  	  uint32_t* deviceTd2,
	  	  uint32_t* deviceTd3,
	  	  uint8_t* deviceTd4
	      )
{
	int shared_memory = threadsPerBlock*4*sizeof(uint32_t);
	__cuda_aes_8b_ptr_decrypt__<<<gridSize,threadsPerBlock,shared_memory,stream>>>(
			m,
			n_blocks,
			offset,
			(const uint32_t*)in,
			(uint32_t*)out,
			(uint32_t*)neigh,
			(uint32_t*)iv,
			expanded_key,
			key_bits,
			deviceTd0,
			deviceTd1,
			deviceTd2,
			deviceTd3,
			deviceTd4
	);
}
