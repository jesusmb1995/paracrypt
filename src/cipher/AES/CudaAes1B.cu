#include "hip/hip_runtime.h"
/*
 *  Copyright (C) 2017 Jesus Martin Berlanga. All Rights Reserved.
 *
 *  This file is part of Paracrypt.
 *
 *  Paracrypt is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  Paracrypt is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with Paracrypt.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

#include "CudaAes1B.cuh"
#include "cuda_logging.cuh"

#define AES_1B_ROUND_KEY(state_pointer, n_state) \
{ \
	if(valid_thread && ti == (n_state)*4+0) { \
		__LOG_TRACE__("p %d: d[%d].0 => 0x%02x",p,d[p]); \
		__LOG_TRACE__("p %d: k[%d].0 => 0x%02x",p,k[4*(n_state)+0]); \
		(state_pointer)[0] = d[p] ^ k[4*(n_state)+0]; \
		__LOG_TRACE__("p %d: state%d.0 => 0x%02x",p,(n_state),(state_pointer)[0]); \
	} \
	else if(valid_thread && ti == 4*(n_state)+1) { \
		__LOG_TRACE__("p %d: d[%d].1 => 0x%02x",p,n_state,d[p]); \
		__LOG_TRACE__("p %d: k[%d].1 => 0x%02x",p,n_state,k[4*(n_state)+1]); \
		(state_pointer)[1] = d[p] ^ k[4*(n_state)+1]; \
		__LOG_TRACE__("p %d: state%d.1 => 0x%02x",p,(n_state),(state_pointer)[1]); \
	} \
	else if(valid_thread && ti == 4*(n_state)+2) { \
		__LOG_TRACE__("p %d: d[%d].2 => 0x%02x",p,n_state,d[p]); \
		__LOG_TRACE__("p %d: k[%d].2 => 0x%02x",p,n_state,k[4*(n_state)+2]); \
		(state_pointer)[2] = d[p] ^ k[4*(n_state)+2]; \
		__LOG_TRACE__("p %d: state%d.2 => 0x%02x",p,(n_state),(state_pointer)[2]); \
	} \
	else if(valid_thread && ti == 4*(n_state)+3) { \
		__LOG_TRACE__("p %d: d[%d].3 => 0x%02x",p,n_state,d[p]); \
		__LOG_TRACE__("p %d: k[%d].3 => 0x%02x",p,n_state,k[4*(n_state)+3]); \
		(state_pointer)[3] = d[p] ^ k[4*(n_state)+3]; \
		__LOG_TRACE__("p %d: state%d.3 => 0x%02x",p,(n_state),(state_pointer)[3]); \
	} \
}

#define AES_1B_PTR_ROUND_KEY(state_pointer, n_state, global_bi_pointer) \
{ \
	if(valid_thread && ti == (n_state)*4+0) { \
		__LOG_TRACE__("p %d: global_bi.0 => 0x%02x",p,(global_bi_pointer)[0]); \
		__LOG_TRACE__("p %d: iv[%d].0 => 0x%02x",p,iv[4*(n_state)+0]); \
		__LOG_TRACE__("p %d: k[%d].0 => 0x%02x",p,k[4*(n_state)+0]); \
		(state_pointer)[0] = (global_bi_pointer)[0] ^ iv[4*(n_state)+0] ^ k[4*(n_state)+0]; \
		__LOG_TRACE__("p %d: state%d.0 => 0x%02x",p,(n_state),(state_pointer)[0]); \
	} \
	else if(valid_thread && ti == 4*(n_state)+1) { \
		__LOG_TRACE__("p %d: global_bi.1 => 0x%02x",p,(global_bi_pointer)[1]); \
		__LOG_TRACE__("p %d: iv[%d].1 => 0x%02x",p,iv[4*(n_state)+1]); \
		__LOG_TRACE__("p %d: k[%d].1 => 0x%02x",p,k[4*(n_state)+1]); \
		(state_pointer)[1] = (global_bi_pointer)[1] ^ iv[4*(n_state)+1] ^ k[4*(n_state)+1]; \
		__LOG_TRACE__("p %d: state%d.1 => 0x%02x",p,(n_state),(state_pointer)[1]); \
	} \
	else if(valid_thread && ti == 4*(n_state)+2) { \
		__LOG_TRACE__("p %d: global_bi.2 => 0x%02x",p,(global_bi_pointer)[2]); \
		__LOG_TRACE__("p %d: iv[%d].2 => 0x%02x",p,iv[4*(n_state)+2]); \
		__LOG_TRACE__("p %d: k[%d].2 => 0x%02x",p,k[4*(n_state)+2]); \
		(state_pointer)[2] = (global_bi_pointer)[2] ^ iv[4*(n_state)+2] ^ k[4*(n_state)+2]; \
		__LOG_TRACE__("p %d: state%d.2 => 0x%02x",p,(n_state),(state_pointer)[2]); \
	} \
	else if(valid_thread && ti == 4*(n_state)+3) { \
		__LOG_TRACE__("p %d: global_bi.3 => 0x%02x",p,(global_bi_pointer)[3]); \
		__LOG_TRACE__("p %d: iv[%d].3 => 0x%02x",p,iv[4*(n_state)+3]); \
		__LOG_TRACE__("p %d: k[%d].3 => 0x%02x",p,k[4*(n_state)+3]); \
		(state_pointer)[3] = (global_bi_pointer)[3] ^ iv[4*(n_state)+3] ^ k[4*(n_state)+3]; \
		__LOG_TRACE__("p %d: state%d.3 => 0x%02x",p,(n_state),(state_pointer)[3]); \
	} \
}

#define AES_1B_CFB_ROUND_KEY(state_pointer, n_state) \
{ \
	if(valid_thread && ti == 4*(n_state)+0) { \
			if(bi == (n_state)*4+0) { \
				__LOG_TRACE__("p %d: iv[%d].0 => 0x%04x",p,iv[4*(n_state)+0]); \
				(state_pointer)[0] = iv[4*(n_state)+0] ^ k[4*(n_state)+0]; \
			} \
			else { \
				if(sti == 0) { \
					int np = (blockIdx.x*16)-16; \
					(state_pointer)[0] = neigh[np+(n_state*4+0)] ^ k[4*(n_state)+0]; \
					__LOG_TRACE__("p %d (bi %d): accessing neighbor at byte %d.",p,bi,np+(n_state*4+0)); \
				} \
				else { \
					(state_pointer)[0] = d[p-16] ^ k[4*(n_state)+0]; \
					__LOG_TRACE__("p %d (bi %d): accessing prev. block.",p,bi); \
				} \
			} \
	} \
	else if(valid_thread && ti == 4*(n_state)+1) { \
			if(bi == (n_state)*4+1) { \
				__LOG_TRACE__("p %d: iv[%d].1 => 0x%04x",p,iv[4*(n_state)+1]); \
				(state_pointer)[1] = iv[4*(n_state)+1] ^ k[4*(n_state)+1]; \
			} \
			else { \
				if(sti == 0) { \
					int np = (blockIdx.x*16)-16; \
					(state_pointer)[1] = neigh[np+(n_state*4+1)] ^ k[4*(n_state)+1]; \
					__LOG_TRACE__("p %d (bi %d): accessing neighbor at byte %d.",p,bi,np+(n_state*4+1)); \
				} \
				else { \
					(state_pointer)[1] = d[p-16] ^ k[4*(n_state)+1]; \
					__LOG_TRACE__("p %d (bi %d): accessing prev. block.",p,bi); \
				} \
			} \
	} \
	else if(valid_thread && ti == 4*(n_state)+2) { \
			if(bi == (n_state)*4+2) { \
				__LOG_TRACE__("p %d: iv[%d].2 => 0x%04x",p,iv[4*(n_state)+2]); \
				(state_pointer)[2] = iv[4*(n_state)+2] ^ k[4*(n_state)+2]; \
			} \
			else { \
				if(sti == 0) { \
					int np = (blockIdx.x*16)-16; \
					(state_pointer)[2] = neigh[np+(n_state*4+2)] ^ k[4*(n_state)+2]; \
					__LOG_TRACE__("p %d (bi %d): accessing neighbor at byte %d.",p,bi,np+(n_state*4+2)); \
				} \
				else { \
					(state_pointer)[2] = d[p-16] ^ k[4*(n_state)+2]; \
					__LOG_TRACE__("p %d (bi %d): accessing prev. block.",p,bi); \
				} \
			} \
	} \
	else if(valid_thread && ti == 4*(n_state)+3) { \
			if(bi == (n_state)*4+3) { \
				__LOG_TRACE__("p %d: iv[%d].3 => 0x%04x",p,iv[4*(n_state)+3]); \
				(state_pointer)[3] = iv[4*(n_state)+3] ^ k[4*(n_state)+3]; \
			} \
			else { \
				if(sti == 0) { \
					int np = (blockIdx.x*16)-16; \
					(state_pointer)[3] = neigh[np+(n_state*4+3)] ^ k[4*(n_state)+3]; \
					__LOG_TRACE__("p %d (bi %d): accessing neighbor at byte %d.",p,bi,np+(n_state*4+3)); \
				} \
				else { \
					(state_pointer)[3] = d[p-16] ^ k[4*(n_state)+3]; \
					__LOG_TRACE__("p %d (bi %d): accessing prev. block.",p,bi); \
				} \
			} \
	} \
}

#define AES_1B_CBC_ROUND(state_pointer, n_state) \
{ \
	if(valid_thread && ti == 4*(n_state)+0) { \
			uint8_t cb0; \
			if(bi == (n_state)*4+0) { \
				__LOG_TRACE__("p %d: iv[%d].0 => 0x%04x",p,iv[4*(n_state)+0]); \
				cb0 = iv[4*(n_state)+0] ^ k[4*(n_state)+0]; \
			} \
			else { \
				if(sti == 0) { \
					int np = (blockIdx.x*16)-16; \
					cb0 = neigh[np+(n_state*4+0)]; \
					__LOG_TRACE__("p %d (bi %d): accessing neighbor at byte %d.",p,bi,np+(n_state*4+0)); \
				} \
				else { \
					cb0 = d[p-16]; \
					__LOG_TRACE__("p %d (bi %d): accessing prev. block.",p,bi); \
				} \
			} \
			(state_pointer)[0] = cb0; \
	} \
	else if(valid_thread && ti == 4*(n_state)+1) { \
		uint8_t cb1; \
		if(bi == (n_state)*4+1) { \
			__LOG_TRACE__("p %d: iv[%d].1=> 0x%04x",p,iv[4*(n_state)+1]); \
			cb1 = iv[4*(n_state)+1] ^ k[4*(n_state)+1]; \
		} \
		else { \
			if(sti == 0) { \
				int np = (blockIdx.x*16)-16; \
				cb1 = neigh[np+(n_state*4+1)]; \
				__LOG_TRACE__("p %d (bi %d): accessing neighbor at byte %d.",p,bi,np+(n_state*4+1)); \
			} \
			else { \
				cb1 = d[p-16]; \
				__LOG_TRACE__("p %d (bi %d): accessing prev. block.",p,bi); \
			} \
		} \
		(state_pointer)[1] = cb1; \
	} \
	else if(valid_thread && ti == 4*(n_state)+2) { \
		uint8_t cb2; \
		if(bi == (n_state)*4+2) { \
			__LOG_TRACE__("p %d: iv[%d].2 => 0x%04x",p,iv[4*(n_state)+2]); \
			cb2 = iv[4*(n_state)+2] ^ k[4*(n_state)+2]; \
		} \
		else { \
			if(sti == 0) { \
				int np = (blockIdx.x*16)-16; \
				cb2 = neigh[np+(n_state*4+2)]; \
				__LOG_TRACE__("p %d (bi %d): accessing neighbor at byte %d.",p,bi,np+(n_state*4+2)); \
			} \
			else { \
				cb2 = d[p-16]; \
				__LOG_TRACE__("p %d (bi %d): accessing prev. block.",p,bi); \
			} \
		} \
		(state_pointer)[2] = cb2; \
	} \
	else if(valid_thread && ti == 4*(n_state)+3) { \
		uint8_t cb3; \
		if(bi == (n_state)*4+3) { \
			__LOG_TRACE__("p %d: iv[%d].3 => 0x%04x",p,iv[4*(n_state)+3]); \
			cb3 = iv[4*(n_state)+3] ^ k[4*(n_state)+3]; \
		} \
		else { \
			if(sti == 0) { \
				int np = (blockIdx.x*16)-16; \
				cb3 = neigh[np+(n_state*4+3)]; \
				__LOG_TRACE__("p %d (bi %d): accessing neighbor at byte %d.",p,bi,np+(n_state*4+3)); \
			} \
			else { \
				cb3 = d[p-16]; \
				__LOG_TRACE__("p %d (bi %d): accessing prev. block.",p,bi); \
			} \
		} \
		(state_pointer)[3] = cb3; \
	} \
}

#define AES_1B_STATE_XOR_INPUT(state_pointer, n_state) \
		if(valid_thread && ti == 4*(n_state)+0) { \
			(state_pointer)[0] ^= d[p]; \
		} \
		else if(valid_thread && ti == 4*(n_state)+1) { \
			(state_pointer)[1] ^= d[p]; \
		} \
		else if(valid_thread && ti == 4*(n_state)+2) { \
			(state_pointer)[2] ^= d[p]; \
		} \
		else if(valid_thread && ti == 4*(n_state)+3) { \
			(state_pointer)[3] ^= d[p]; \
		} \

#define AES_1B_EXPORT_STATE(state_pointer, n_state) \
	if(valid_thread && ti == 4*(n_state)+0) { \
		out[p] = (state_pointer)[0]; \
	} \
	else if(valid_thread && ti == 4*(n_state)+1) { \
		out[p] = (state_pointer)[1]; \
	} \
	else if(valid_thread && ti == 4*(n_state)+2) { \
		out[p] = (state_pointer)[2]; \
	} \
	else if(valid_thread && ti == 4*(n_state)+3) { \
		out[p] = (state_pointer)[3]; \
	} \

#define AES_1B_ENCRYPT_ROUND(store0_ptr,store1_ptr,store2_ptr,store3_ptr,s0_ptr, s1_ptr, s2_ptr, s3_ptr, round_number) \
{ \
	/* S0 = ... */ \
	if(valid_thread && ti == 0) { \
		(store0_ptr)[0] = T0[(s0_ptr)[0]*4+0] ^ T1[(s1_ptr)[1]*4+0] ^ T2[(s2_ptr)[2]*4+0] ^ T3[(s3_ptr)[3]*4+0] ^ k[4*((round_number)*4)+0]; \
		__LOG_TRACE__("p %d: state0.0 => 0x%02x",p,(store0_ptr)[0]); \
	} \
	else if(valid_thread && ti == 1) { \
		(store0_ptr)[1] = T0[(s0_ptr)[0]*4+1] ^ T1[(s1_ptr)[1]*4+1] ^ T2[(s2_ptr)[2]*4+1] ^ T3[(s3_ptr)[3]*4+1] ^ k[4*((round_number)*4)+1]; \
		__LOG_TRACE__("p %d: state0.1 => 0x%02x",p,(store0_ptr)[1]); \
	} \
	else if(valid_thread && ti == 2) { \
		(store0_ptr)[2] = T0[(s0_ptr)[0]*4+2] ^ T1[(s1_ptr)[1]*4+2] ^ T2[(s2_ptr)[2]*4+2] ^ T3[(s3_ptr)[3]*4+2] ^ k[4*((round_number)*4)+2]; \
		__LOG_TRACE__("p %d: state0.3 => 0x%02x",p,(store0_ptr)[2]); \
	} \
	else if(valid_thread && ti == 3) { \
		(store0_ptr)[3] = T0[(s0_ptr)[0]*4+3] ^ T1[(s1_ptr)[1]*4+3] ^ T2[(s2_ptr)[2]*4+3] ^ T3[(s3_ptr)[3]*4+3] ^ k[4*((round_number)*4)+3]; \
		__LOG_TRACE__("p %d: state0.3 => 0x%02x",p,(store0_ptr)[3]); \
	} \
	\
	/* S1 = ... */ \
	else if(valid_thread && ti == 4) { \
		(store1_ptr)[0] = T0[(s1_ptr)[0]*4+0] ^ T1[(s2_ptr)[1]*4+0] ^ T2[(s3_ptr)[2]*4+0] ^ T3[(s0_ptr)[3]*4+0] ^ k[4*((round_number)*4)+4]; \
		__LOG_TRACE__("p %d: state0.0 => 0x%02x",p,(store1_ptr)[0]); \
	} \
	else if(valid_thread && ti == 5) { \
		(store1_ptr)[1] = T0[(s1_ptr)[0]*4+1] ^ T1[(s2_ptr)[1]*4+1] ^ T2[(s3_ptr)[2]*4+1] ^ T3[(s0_ptr)[3]*4+1] ^ k[4*((round_number)*4)+5]; \
		__LOG_TRACE__("p %d: state0.1 => 0x%02x",p,(store1_ptr)[1]); \
	} \
	else if(valid_thread && ti == 6) { \
		(store1_ptr)[2] = T0[(s1_ptr)[0]*4+2] ^ T1[(s2_ptr)[1]*4+2] ^ T2[(s3_ptr)[2]*4+2] ^ T3[(s0_ptr)[3]*4+2] ^ k[4*((round_number)*4)+6]; \
		__LOG_TRACE__("p %d: state0.3 => 0x%02x",p,(store1_ptr)[2]); \
	} \
	else if(valid_thread && ti == 7) { \
		(store1_ptr)[3] = T0[(s1_ptr)[0]*4+3] ^ T1[(s2_ptr)[1]*4+3] ^ T2[(s3_ptr)[2]*4+3] ^ T3[(s0_ptr)[3]*4+3] ^ k[4*((round_number)*4)+7]; \
		__LOG_TRACE__("p %d: state0.3 => 0x%02x",p,(store1_ptr)[3]); \
	} \
	\
	/* S2 = ... */ \
	else if(valid_thread && ti == 8) { \
		(store2_ptr)[0] = T0[(s2_ptr)[0]*4+0] ^ T1[(s3_ptr)[1]*4+0] ^ T2[(s0_ptr)[2]*4+0] ^ T3[(s1_ptr)[3]*4+0] ^ k[4*((round_number)*4)+8]; \
		__LOG_TRACE__("p %d: state0.0 => 0x%02x",p,(store2_ptr)[0]); \
	} \
	else if(valid_thread && ti == 9) { \
		(store2_ptr)[1] = T0[(s2_ptr)[0]*4+1] ^ T1[(s3_ptr)[1]*4+1] ^ T2[(s0_ptr)[2]*4+1] ^ T3[(s1_ptr)[3]*4+1] ^ k[4*((round_number)*4)+9]; \
		__LOG_TRACE__("p %d: state0.1 => 0x%02x",p,(store2_ptr)[1]); \
	} \
	else if(valid_thread && ti == 10) { \
		(store2_ptr)[2] = T0[(s2_ptr)[0]*4+2] ^ T1[(s3_ptr)[1]*4+2] ^ T2[(s0_ptr)[2]*4+2] ^ T3[(s1_ptr)[3]*4+2] ^ k[4*((round_number)*4)+10]; \
		__LOG_TRACE__("p %d: state0.3 => 0x%02x",p,(store2_ptr)[2]); \
	} \
	else if(valid_thread && ti == 11) { \
		(store2_ptr)[3] = T0[(s2_ptr)[0]*4+3] ^ T1[(s3_ptr)[1]*4+3] ^ T2[(s0_ptr)[2]*4+3] ^ T3[(s1_ptr)[3]*4+3] ^ k[4*((round_number)*4)+11]; \
		__LOG_TRACE__("p %d: state0.3 => 0x%02x",p,(store2_ptr)[3]); \
	} \
	\
	/* S3 = ... */ \
	else if(valid_thread && ti == 12) { \
		(store3_ptr)[0] = T0[(s3_ptr)[0]*4+0] ^ T1[(s0_ptr)[1]*4+0] ^ T2[(s1_ptr)[2]*4+0] ^ T3[(s2_ptr)[3]*4+0] ^ k[4*((round_number)*4)+12]; \
		__LOG_TRACE__("p %d: state0.0 => 0x%02x",p,(store2_ptr)[0]); \
	} \
	else if(valid_thread && ti == 13) { \
		(store3_ptr)[1] = T0[(s3_ptr)[0]*4+1] ^ T1[(s0_ptr)[1]*4+1] ^ T2[(s1_ptr)[2]*4+1] ^ T3[(s2_ptr)[3]*4+1] ^ k[4*((round_number)*4)+13]; \
		__LOG_TRACE__("p %d: state0.1 => 0x%02x",p,(store2_ptr)[1]); \
	} \
	else if(valid_thread && ti == 14) { \
		(store3_ptr)[2] = T0[(s3_ptr)[0]*4+2] ^ T1[(s0_ptr)[1]*4+2] ^ T2[(s1_ptr)[2]*4+2] ^ T3[(s2_ptr)[3]*4+2] ^ k[4*((round_number)*4)+14]; \
		__LOG_TRACE__("p %d: state0.3 => 0x%02x",p,(store2_ptr)[2]); \
	} \
	else if(valid_thread && ti == 15) { \
		(store3_ptr)[3] = T0[(s3_ptr)[0]*4+3] ^ T1[(s0_ptr)[1]*4+3] ^ T2[(s1_ptr)[2]*4+3] ^ T3[(s2_ptr)[3]*4+3] ^ k[4*((round_number)*4)+15]; \
		__LOG_TRACE__("p %d: state0.3 => 0x%02x",p,(store2_ptr)[3]); \
	} \
}

#define AES_1B_DECRYPT_ROUND(store0_ptr,store1_ptr,store2_ptr,store3_ptr,s0_ptr, s1_ptr, s2_ptr, s3_ptr, round_number) \
{ \
	/* S0 = ... */ \
	if(valid_thread && ti == 0) { \
		(store0_ptr)[0] = T0[(s0_ptr)[0]*4+0] ^ T1[(s3_ptr)[1]*4+0] ^ T2[(s2_ptr)[2]*4+0] ^ T3[(s1_ptr)[3]*4+0] ^ k[4*((round_number)*4)+0]; \
		__LOG_TRACE__("p %d: state0.0 => 0x%02x",p,(store0_ptr)[0]); \
	} \
	else if(valid_thread && ti == 1) { \
		(store0_ptr)[1] = T0[(s0_ptr)[0]*4+1] ^ T1[(s3_ptr)[1]*4+1] ^ T2[(s2_ptr)[2]*4+1] ^ T3[(s1_ptr)[3]*4+1] ^ k[4*((round_number)*4)+1]; \
		__LOG_TRACE__("p %d: state0.1 => 0x%02x",p,(store0_ptr)[1]); \
	} \
	else if(valid_thread && ti == 2) { \
		(store0_ptr)[2] = T0[(s0_ptr)[0]*4+2] ^ T1[(s3_ptr)[1]*4+2] ^ T2[(s2_ptr)[2]*4+2] ^ T3[(s1_ptr)[3]*4+2] ^ k[4*((round_number)*4)+2]; \
		__LOG_TRACE__("p %d: state0.3 => 0x%02x",p,(store0_ptr)[2]); \
	} \
	else if(valid_thread && ti == 3) { \
		(store0_ptr)[3] = T0[(s0_ptr)[0]*4+3] ^ T1[(s3_ptr)[1]*4+3] ^ T2[(s2_ptr)[2]*4+3] ^ T3[(s1_ptr)[3]*4+3] ^ k[4*((round_number)*4)+3]; \
		__LOG_TRACE__("p %d: state0.3 => 0x%02x",p,(store0_ptr)[3]); \
	} \
	\
	/* S1 = ... */ \
	else if(valid_thread && ti == 4) { \
		(store1_ptr)[0] = T0[(s1_ptr)[0]*4+0] ^ T1[(s0_ptr)[1]*4+0] ^ T2[(s3_ptr)[2]*4+0] ^ T3[(s2_ptr)[3]*4+0] ^ k[4*((round_number)*4)+4]; \
		__LOG_TRACE__("p %d: state0.0 => 0x%02x",p,(store1_ptr)[0]); \
	} \
	else if(valid_thread && ti == 5) { \
		(store1_ptr)[1] = T0[(s1_ptr)[0]*4+1] ^ T1[(s0_ptr)[1]*4+1] ^ T2[(s3_ptr)[2]*4+1] ^ T3[(s2_ptr)[3]*4+1] ^ k[4*((round_number)*4)+5]; \
		__LOG_TRACE__("p %d: state0.1 => 0x%02x",p,(store1_ptr)[1]); \
	} \
	else if(valid_thread && ti == 6) { \
		(store1_ptr)[2] = T0[(s1_ptr)[0]*4+2] ^ T1[(s0_ptr)[1]*4+2] ^ T2[(s3_ptr)[2]*4+2] ^ T3[(s2_ptr)[3]*4+2] ^ k[4*((round_number)*4)+6]; \
		__LOG_TRACE__("p %d: state0.3 => 0x%02x",p,(store1_ptr)[2]); \
	} \
	else if(valid_thread && ti == 7) { \
		(store1_ptr)[3] = T0[(s1_ptr)[0]*4+3] ^ T1[(s0_ptr)[1]*4+3] ^ T2[(s3_ptr)[2]*4+3] ^ T3[(s2_ptr)[3]*4+3] ^ k[4*((round_number)*4)+7]; \
		__LOG_TRACE__("p %d: state0.3 => 0x%02x",p,(store1_ptr)[3]); \
	} \
	\
	/* S2 = ... */ \
	else if(valid_thread && ti == 8) { \
		(store2_ptr)[0] = T0[(s2_ptr)[0]*4+0] ^ T1[(s1_ptr)[1]*4+0] ^ T2[(s0_ptr)[2]*4+0] ^ T3[(s3_ptr)[3]*4+0] ^ k[4*((round_number)*4)+8]; \
		__LOG_TRACE__("p %d: state0.0 => 0x%02x",p,(store2_ptr)[0]); \
	} \
	else if(valid_thread && ti == 9) { \
		(store2_ptr)[1] = T0[(s2_ptr)[0]*4+1] ^ T1[(s1_ptr)[1]*4+1] ^ T2[(s0_ptr)[2]*4+1] ^ T3[(s3_ptr)[3]*4+1] ^ k[4*((round_number)*4)+9]; \
		__LOG_TRACE__("p %d: state0.1 => 0x%02x",p,(store2_ptr)[1]); \
	} \
	else if(valid_thread && ti == 10) { \
		(store2_ptr)[2] = T0[(s2_ptr)[0]*4+2] ^ T1[(s1_ptr)[1]*4+2] ^ T2[(s0_ptr)[2]*4+2] ^ T3[(s3_ptr)[3]*4+2] ^ k[4*((round_number)*4)+10]; \
		__LOG_TRACE__("p %d: state0.3 => 0x%02x",p,(store2_ptr)[2]); \
	} \
	else if(valid_thread && ti == 11) { \
		(store2_ptr)[3] = T0[(s2_ptr)[0]*4+3] ^ T1[(s1_ptr)[1]*4+3] ^ T2[(s0_ptr)[2]*4+3] ^ T3[(s3_ptr)[3]*4+3] ^ k[4*((round_number)*4)+11]; \
		__LOG_TRACE__("p %d: state0.3 => 0x%02x",p,(store2_ptr)[3]); \
	} \
	\
	/* S3 = ... */ \
	else if(valid_thread && ti == 12) { \
		(store3_ptr)[0] = T0[(s3_ptr)[0]*4+0] ^ T1[(s2_ptr)[1]*4+0] ^ T2[(s1_ptr)[2]*4+0] ^ T3[(s0_ptr)[3]*4+0] ^ k[4*((round_number)*4)+12]; \
		__LOG_TRACE__("p %d: state0.0 => 0x%02x",p,(store2_ptr)[0]); \
	} \
	else if(valid_thread && ti == 13) { \
		(store3_ptr)[1] = T0[(s3_ptr)[0]*4+1] ^ T1[(s2_ptr)[1]*4+1] ^ T2[(s1_ptr)[2]*4+1] ^ T3[(s0_ptr)[3]*4+1] ^ k[4*((round_number)*4)+13]; \
		__LOG_TRACE__("p %d: state0.1 => 0x%02x",p,(store2_ptr)[1]); \
	} \
	else if(valid_thread && ti == 14) { \
		(store3_ptr)[2] = T0[(s3_ptr)[0]*4+2] ^ T1[(s2_ptr)[1]*4+2] ^ T2[(s1_ptr)[2]*4+2] ^ T3[(s0_ptr)[3]*4+2] ^ k[4*((round_number)*4)+14]; \
		__LOG_TRACE__("p %d: state0.3 => 0x%02x",p,(store2_ptr)[2]); \
	} \
	else if(valid_thread && ti == 15) { \
		(store3_ptr)[3] = T0[(s3_ptr)[0]*4+3] ^ T1[(s2_ptr)[1]*4+3] ^ T2[(s1_ptr)[2]*4+3] ^ T3[(s0_ptr)[3]*4+3] ^ k[4*((round_number)*4)+15]; \
		__LOG_TRACE__("p %d: state0.3 => 0x%02x",p,(store2_ptr)[3]); \
	} \
}

__global__ void __cuda_aes_1b_encrypt__(
		  const paracrypt::BlockCipher::Mode m,
		  unsigned int n,
		  uint32_t offset,
		  const uint8_t* d,
		  uint8_t* out,
		  uint8_t* neigh,
		  uint8_t* iv,
	  	  uint8_t* k,
		  const int key_bits,
	  	  uint8_t* T0,
	  	  uint8_t* T1,
	  	  uint8_t* T2,
	  	  uint8_t* T3
    )
{
	// Each block has its own shared memory
	// We have an state for each two threads
	extern __shared__ uint32_t state[];

	unsigned int bi = ((blockIdx.x * blockDim.x) + threadIdx.x); // byte index
	const int s_size = blockDim.x/16;
	//__LOG_TRACE__("s_size => %d", s_size);
	uint32_t* s0 = state           ;
	uint32_t* s1 = state+(  s_size);
	uint32_t* s2 = state+(2*s_size);
	uint32_t* s3 = state+(3*s_size);
	uint32_t* t0 = state+(4*s_size);
	uint32_t* t1 = state+(5*s_size);
	uint32_t* t2 = state+(6*s_size);
	uint32_t* t3 = state+(7*s_size);

	unsigned int p = bi;
	uint32_t sti = threadIdx.x/16; //state index
	int ti = threadIdx.x%16;
	int valid_thread = bi < n*16;
	unsigned int extra_rounds = 0;

	uint8_t* s0p = (uint8_t*) &s0[sti];
	uint8_t* s1p = (uint8_t*) &s1[sti];
	uint8_t* s2p = (uint8_t*) &s2[sti];
	uint8_t* s3p = (uint8_t*) &s3[sti];
	uint8_t* t0p = (uint8_t*) &t0[sti];
	uint8_t* t1p = (uint8_t*) &t1[sti];
	uint8_t* t2p = (uint8_t*) &t2[sti];
	uint8_t* t3p = (uint8_t*) &t3[sti];

#if defined(DEBUG) && defined(DEVEL)
	if(valid_thread) {
    	__LOG_TRACE__("p %d: threadIx.x => %d",p,threadIdx.x);
    	__LOG_TRACE__("p %d: ti => %d",p,ti);
    }
#endif

	/*
	 * map byte array block to cipher state
	 * and add initial round key:
	 */
	if(m == paracrypt::BlockCipher::CTR || m == paracrypt::BlockCipher::GCM) {
		uint32_t global_bi = offset+sti;
		uint8_t* global_bi_pointer = (uint8_t*) &global_bi;
		AES_1B_PTR_ROUND_KEY(s0p, 0, global_bi_pointer);
		AES_1B_PTR_ROUND_KEY(s1p, 1, global_bi_pointer);
		AES_1B_PTR_ROUND_KEY(s2p, 2, global_bi_pointer);
		AES_1B_PTR_ROUND_KEY(s3p, 3, global_bi_pointer);
	}
	else if(m == paracrypt::BlockCipher::CFB) {
		AES_1B_CFB_ROUND_KEY(s0p,0);
		AES_1B_CFB_ROUND_KEY(s1p,1);
		AES_1B_CFB_ROUND_KEY(s2p,2);
		AES_1B_CFB_ROUND_KEY(s3p,3);
	}
	else {
		AES_1B_ROUND_KEY(s0p,0);
		AES_1B_ROUND_KEY(s1p,1);
		AES_1B_ROUND_KEY(s2p,2);
		AES_1B_ROUND_KEY(s3p,3);
	}

	// 8 rounds
	#pragma unroll
	for(int r2 = 1; r2 <= 4; r2++) {
		__syncthreads();
		AES_1B_ENCRYPT_ROUND(t0p,t1p,t2p,t3p,s0p,s1p,s2p,s3p,r2*2-1);
		__syncthreads();
		AES_1B_ENCRYPT_ROUND(s0p,s1p,s2p,s3p,t0p,t1p,t2p,t3p,r2*2);
	}

	// +2 rounds
	if(key_bits >= 192) {
		extra_rounds = 2;
		__syncthreads();
		AES_1B_ENCRYPT_ROUND(t0p,t1p,t2p,t3p,s0p,s1p,s2p,s3p,9);
		__syncthreads();
		AES_1B_ENCRYPT_ROUND(s0p,s1p,s2p,s3p,t0p,t1p,t2p,t3p,10);

		// +2 rounds
		if(key_bits == 256) {
			extra_rounds = 4;
			__syncthreads();
			AES_1B_ENCRYPT_ROUND(t0p,t1p,t2p,t3p,s0p,s1p,s2p,s3p,11);
			__syncthreads();
			AES_1B_ENCRYPT_ROUND(s0p,s1p,s2p,s3p,t0p,t1p,t2p,t3p,12);
		}
	}

	__syncthreads();
	AES_1B_ENCRYPT_ROUND(t0p,t1p,t2p,t3p,s0p,s1p,s2p,s3p,9+extra_rounds);

	__syncthreads();
	// last round
	/* S0 = ... */
	if(valid_thread && ti == 0) {
		s0p[0] = T2[t0p[0]*4+0] ^ k[4*((10+extra_rounds)*4)+0];
		__LOG_TRACE__("p %d: state0.0 => 0x%02x",p,s0p[0]);
	}
	else if(valid_thread && ti == 1) {
		s0p[1] = T3[t1p[1]*4+1] ^ k[4*((10+extra_rounds)*4)+1];
		__LOG_TRACE__("p %d: state0.1 => 0x%02x",p,s0p[1]);
	}
	else if(valid_thread && ti == 2) {
		s0p[2] = T0[t2p[2]*4+2] ^ k[4*((10+extra_rounds)*4)+2];
		__LOG_TRACE__("p %d: state0.2 => 0x%02x",p,s0p[2]);
	}
	else if(valid_thread && ti == 3) {
		s0p[3] = T1[t3p[3]*4+3] ^ k[4*((10+extra_rounds)*4)+3];
		__LOG_TRACE__("p %d: state0.3 => 0x%02x",p,s0p[3]);
	}
	/* S1 = ... */
	else if(valid_thread && ti == 4) {
		s1p[0] = T2[t1p[0]*4+0] ^ k[4*((10+extra_rounds)*4)+4];
		__LOG_TRACE__("p %d: state1.0 => 0x%02x",p,s1p[0]);
	}
	else if(valid_thread && ti == 5) {
		s1p[1] = T3[t2p[1]*4+1] ^ k[4*((10+extra_rounds)*4)+5];
		__LOG_TRACE__("p %d: state1.1 => 0x%02x",p,s1p[1]);
	}
	else if(valid_thread && ti == 6) {
		s1p[2] = T0[t3p[2]*4+2] ^ k[4*((10+extra_rounds)*4)+6];
		__LOG_TRACE__("p %d: state1.2 => 0x%02x",p,s1p[2]);
	}
	else if(valid_thread && ti == 7) {
		s1p[3] = T1[t0p[3]*4+3] ^ k[4*((10+extra_rounds)*4)+7];
		__LOG_TRACE__("p %d: state1.3 => 0x%02x",p,s1p[3]);
	}
	/* S2 = ... */
	else if(valid_thread && ti == 8) {
		s2p[0] = T2[t2p[0]*4+0] ^ k[4*((10+extra_rounds)*4)+8];
		__LOG_TRACE__("p %d: state2.0 => 0x%02x",p,s2p[0]);
	}
	else if(valid_thread && ti == 9) {
		s2p[1] = T3[t3p[1]*4+1] ^ k[4*((10+extra_rounds)*4)+9];
		__LOG_TRACE__("p %d: state2.1 => 0x%02x",p,s2p[1]);
	}
	else if(valid_thread && ti == 10) {
		s2p[2] = T0[t0p[2]*4+2] ^ k[4*((10+extra_rounds)*4)+10];
		__LOG_TRACE__("p %d: state2.2 => 0x%02x",p,s2p[2]);
	}
	else if(valid_thread && ti == 11) {
		s2p[3] = T1[t1p[3]*4+3] ^ k[4*((10+extra_rounds)*4)+11];
		__LOG_TRACE__("p %d: state2.3 => 0x%02x",p,s2p[3]);
	}
	/* S3 = ... */
	else if(valid_thread && ti == 12) {
		s3p[0] = T2[t3p[0]*4+0] ^ k[4*((10+extra_rounds)*4)+12];
		__LOG_TRACE__("p %d: state3.0 => 0x%02x",p,s3p[0]);
	}
	else if(valid_thread && ti == 13) {
		s3p[1] = T3[t0p[1]*4+1] ^ k[4*((10+extra_rounds)*4)+13];
		__LOG_TRACE__("p %d: state3.1 => 0x%02x",p,s3p[1]);
	}
	else if(valid_thread && ti == 14) {
		s3p[2] = T0[t1p[2]*4+2] ^ k[4*((10+extra_rounds)*4)+14];
		__LOG_TRACE__("p %d: state3.2 => 0x%02x",p,s3p[2]);
	}
	else if(valid_thread && ti == 15) {
		s3p[3] = T1[t2p[3]*4+3] ^ k[4*((10+extra_rounds)*4)+15];
		__LOG_TRACE__("p %d: state3.3 => 0x%02x",p,s3p[3]);
	}

	if(		   m == paracrypt::BlockCipher::CTR
			|| m == paracrypt::BlockCipher::GCM
			|| m == paracrypt::BlockCipher::CFB
	){
		AES_1B_STATE_XOR_INPUT(s0p,0);
		AES_1B_STATE_XOR_INPUT(s1p,1);
		AES_1B_STATE_XOR_INPUT(s2p,2);
		AES_1B_STATE_XOR_INPUT(s3p,3);
	}

	// Save results
	AES_1B_EXPORT_STATE(s0p,0);
	AES_1B_EXPORT_STATE(s1p,1);
	AES_1B_EXPORT_STATE(s2p,2);
	AES_1B_EXPORT_STATE(s3p,3);
}

__global__ void __cuda_aes_1b_decrypt__(
		const paracrypt::BlockCipher::Mode m,
		unsigned int n,
		unsigned int offset,
		const uint8_t* d,
		uint8_t* out,
		uint8_t* neigh,
		uint8_t* iv,
		uint8_t* k,
		const int key_bits,
	  	uint8_t* T0,
	  	uint8_t* T1,
	  	uint8_t* T2,
	  	uint8_t* T3,
	  	uint8_t* T4
    )
{
	// Each block has its own shared memory
	// We have an state for each two threads
	extern __shared__ uint32_t state[];

	unsigned int bi = ((blockIdx.x * blockDim.x) + threadIdx.x); // byte index
	const int s_size = blockDim.x/16;
	//__LOG_TRACE__("s_size => %d", s_size);
	uint32_t* s0 = state           ;
	uint32_t* s1 = state+(  s_size);
	uint32_t* s2 = state+(2*s_size);
	uint32_t* s3 = state+(3*s_size);
	uint32_t* t0 = state+(4*s_size);
	uint32_t* t1 = state+(5*s_size);
	uint32_t* t2 = state+(6*s_size);
	uint32_t* t3 = state+(7*s_size);

	unsigned int p = bi;
	unsigned int sti = threadIdx.x/16; //state index
	int ti = threadIdx.x%16; // thread index: 16 threads per cipher-block
	int valid_thread = bi < n*16;
	unsigned int extra_rounds = 0;

	uint8_t* s0p = (uint8_t*) &s0[sti];
	uint8_t* s1p = (uint8_t*) &s1[sti];
	uint8_t* s2p = (uint8_t*) &s2[sti];
	uint8_t* s3p = (uint8_t*) &s3[sti];
	uint8_t* t0p = (uint8_t*) &t0[sti];
	uint8_t* t1p = (uint8_t*) &t1[sti];
	uint8_t* t2p = (uint8_t*) &t2[sti];
	uint8_t* t3p = (uint8_t*) &t3[sti];

#if defined(DEBUG) && defined(DEVEL)
	if(valid_thread) {
    	__LOG_TRACE__("p %d: threadIx.x => %d",p,threadIdx.x);
    	__LOG_TRACE__("p %d: ti => %d",p,ti);
    }
#endif

	/*
	 * map byte array block to cipher state
	 * and add initial round key:
	 */
	AES_1B_ROUND_KEY(s0p,0);
	AES_1B_ROUND_KEY(s1p,1);
	AES_1B_ROUND_KEY(s2p,2);
	AES_1B_ROUND_KEY(s3p,3);

	// 8 rounds
	#pragma unroll
	for(int r2 = 1; r2 <= 4; r2++) {
		__syncthreads();
		AES_1B_DECRYPT_ROUND(t0p,t1p,t2p,t3p,s0p,s1p,s2p,s3p,r2*2-1);
		__syncthreads();
		AES_1B_DECRYPT_ROUND(s0p,s1p,s2p,s3p,t0p,t1p,t2p,t3p,r2*2);
	}

	// +2 rounds
	if(key_bits >= 192) {
		extra_rounds = 2;
		__syncthreads();
		AES_1B_DECRYPT_ROUND(t0p,t1p,t2p,t3p,s0p,s1p,s2p,s3p,9);
		__syncthreads();
		AES_1B_DECRYPT_ROUND(s0p,s1p,s2p,s3p,t0p,t1p,t2p,t3p,10);

		// +2 rounds
		if(key_bits == 256) {
			extra_rounds = 4;
			__syncthreads();
			AES_1B_DECRYPT_ROUND(t0p,t1p,t2p,t3p,s0p,s1p,s2p,s3p,11);
			__syncthreads();
			AES_1B_DECRYPT_ROUND(s0p,s1p,s2p,s3p,t0p,t1p,t2p,t3p,12);
		}
	}

	__syncthreads();
	AES_1B_DECRYPT_ROUND(t0p,t1p,t2p,t3p,s0p,s1p,s2p,s3p,9+extra_rounds);

	__syncthreads();
	// last round
	/* S0 = ... */
	if(valid_thread && ti == 0) {
		s0p[0] = T4[t0p[0]] ^ k[4*((10+extra_rounds)*4)+0];
		__LOG_TRACE__("p %d: state0.0 => 0x%02x",p,s0p[0]);
	}
	else if(valid_thread && ti == 1) {
		s0p[1] = T4[t3p[1]] ^ k[4*((10+extra_rounds)*4)+1];
		__LOG_TRACE__("p %d: state0.1 => 0x%02x",p,s0p[1]);
	}
	else if(valid_thread && ti == 2) {
		s0p[2] = T4[t2p[2]] ^ k[4*((10+extra_rounds)*4)+2];
		__LOG_TRACE__("p %d: state0.2 => 0x%02x",p,s0p[2]);
	}
	else if(valid_thread && ti == 3) {
		s0p[3] = T4[t1p[3]] ^ k[4*((10+extra_rounds)*4)+3];
		__LOG_TRACE__("p %d: state0.3 => 0x%02x",p,s0p[3]);
	}
	/* S1 = ... */
	else if(valid_thread && ti == 4) {
		s1p[0] = T4[t1p[0]] ^ k[4*((10+extra_rounds)*4)+4];
		__LOG_TRACE__("p %d: state1.0 => 0x%02x",p,s1p[0]);
	}
	else if(valid_thread && ti == 5) {
		s1p[1] = T4[t0p[1]] ^ k[4*((10+extra_rounds)*4)+5];
		__LOG_TRACE__("p %d: state1.1 => 0x%02x",p,s1p[1]);
	}
	else if(valid_thread && ti == 6) {
		s1p[2] = T4[t3p[2]] ^ k[4*((10+extra_rounds)*4)+6];
		__LOG_TRACE__("p %d: state1.2 => 0x%02x",p,s1p[2]);
	}
	else if(valid_thread && ti == 7) {
		s1p[3] = T4[t2p[3]] ^ k[4*((10+extra_rounds)*4)+7];
		__LOG_TRACE__("p %d: state1.3 => 0x%02x",p,s1p[3]);
	}
	/* S2 = ... */
	else if(valid_thread && ti == 8) {
		s2p[0] = T4[t2p[0]] ^ k[4*((10+extra_rounds)*4)+8];
		__LOG_TRACE__("p %d: state2.0 => 0x%02x",p,s2p[0]);
	}
	else if(valid_thread && ti == 9) {
		s2p[1] = T4[t1p[1]] ^ k[4*((10+extra_rounds)*4)+9];
		__LOG_TRACE__("p %d: state2.1 => 0x%02x",p,s2p[1]);
	}
	else if(valid_thread && ti == 10) {
		s2p[2] = T4[t0p[2]] ^ k[4*((10+extra_rounds)*4)+10];
		__LOG_TRACE__("p %d: state2.2 => 0x%02x",p,s2p[2]);
	}
	else if(valid_thread && ti == 11) {
		s2p[3] = T4[t3p[3]] ^ k[4*((10+extra_rounds)*4)+11];
		__LOG_TRACE__("p %d: state2.3 => 0x%02x",p,s2p[3]);
	}
	/* S3 = ... */
	else if(valid_thread && ti == 12) {
		s3p[0] = T4[t3p[0]] ^ k[4*((10+extra_rounds)*4)+12];
		__LOG_TRACE__("p %d: state3.0 => 0x%02x",p,s3p[0]);
	}
	else if(valid_thread && ti == 13) {
		s3p[1] = T4[t2p[1]] ^ k[4*((10+extra_rounds)*4)+13];
		__LOG_TRACE__("p %d: state3.1 => 0x%02x",p,s3p[1]);
	}
	else if(valid_thread && ti == 14) {
		s3p[2] = T4[t1p[2]] ^ k[4*((10+extra_rounds)*4)+14];
		__LOG_TRACE__("p %d: state3.2 => 0x%02x",p,s3p[2]);
	}
	else if(valid_thread && ti == 15) { \
		s3p[3] = T4[t0p[3]] ^ k[4*((10+extra_rounds)*4)+15];
		__LOG_TRACE__("p %d: state3.3 => 0x%02x",p,s3p[3]);
	}

	if(m == paracrypt::BlockCipher::CBC) {
		AES_1B_CBC_ROUND(s0p,0);
		AES_1B_CBC_ROUND(s1p,1);
		AES_1B_CBC_ROUND(s2p,2);
		AES_1B_CBC_ROUND(s3p,3);
		__syncthreads();
	}

	// Save results
	AES_1B_EXPORT_STATE(s0p,0);
	AES_1B_EXPORT_STATE(s1p,1);
	AES_1B_EXPORT_STATE(s2p,2);
	AES_1B_EXPORT_STATE(s3p,3);
}

void cuda_aes_1b_encrypt(
		  paracrypt::BlockCipher::Mode m,
	  	  int gridSize,
	  	  int threadsPerBlock,
	  	  hipStream_t stream,
	  	  unsigned int n_blocks,
	  	  uint32_t offset,
	  	  unsigned char* in,
	  	  unsigned char* out,
	  	  unsigned char* neigh,
	  	  unsigned char* iv,
	  	  uint32_t* expanded_key,
	  	  int key_bits,
	  	  uint32_t* deviceTe0,
	  	  uint32_t* deviceTe1,
	  	  uint32_t* deviceTe2,
	  	  uint32_t* deviceTe3
	      )
{
	int shared_memory = threadsPerBlock*sizeof(uint32_t);
	__cuda_aes_1b_encrypt__<<<gridSize,threadsPerBlock,shared_memory,stream>>>(//*2>>>(
			m,
			n_blocks,
			offset,
			(const uint8_t*)in,
			(uint8_t*)out,
			(uint8_t*)neigh,
			(uint8_t*)iv,
			(uint8_t*)expanded_key,
			key_bits,
	   		(uint8_t*)deviceTe0,
	   		(uint8_t*)deviceTe1,
	   		(uint8_t*)deviceTe2,
	   		(uint8_t*)deviceTe3
	);
}

void cuda_aes_1b_decrypt(
		  paracrypt::BlockCipher::Mode m,
	  	  int gridSize,
	  	  int threadsPerBlock,
	  	  hipStream_t stream,
	  	  unsigned int n_blocks,
	  	  unsigned int offset,
	  	  unsigned char* in,
	  	  unsigned char* out,
	  	  unsigned char* neigh,
	  	  unsigned char* iv,
	  	  uint32_t* expanded_key,
	  	  int key_bits,
	  	  uint32_t* deviceTd0,
	  	  uint32_t* deviceTd1,
	  	  uint32_t* deviceTd2,
	  	  uint32_t* deviceTd3,
	  	  uint8_t* deviceTd4
	      )
{
	int shared_memory = threadsPerBlock*sizeof(uint32_t);
	__cuda_aes_1b_decrypt__<<<gridSize,threadsPerBlock,shared_memory,stream>>>(
			m,
			n_blocks,
			offset,
			(const uint8_t*)in,
			(uint8_t*)out,
			(uint8_t*)neigh,
			(uint8_t*)iv,
			(uint8_t*)expanded_key,
			key_bits,
			(uint8_t*)deviceTd0,
			(uint8_t*)deviceTd1,
			(uint8_t*)deviceTd2,
			(uint8_t*)deviceTd3,
	   		deviceTd4
	);
}
