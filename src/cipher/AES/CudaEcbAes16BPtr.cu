#include "hip/hip_runtime.h"
/*
 *  Copyright (C) 2017 Jesus Martin Berlanga. All Rights Reserved.
 *
 *  This file is part of Paracrypt.
 *
 *  Paracrypt is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  Paracrypt is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with Paracrypt.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

#include "CudaEcbAes16BPtr.cuh"

__global__ void __cuda_ecb_aes_16b_ptr_encrypt__(
		  int n,
		  uint32_t* d,
	  	  uint32_t* k,
	  	  int key_bits,
	  	  uint32_t* T0,
	  	  uint32_t* T1,
	  	  uint32_t* T2,
	  	  uint32_t* T3
    )
{
	int bi = ((blockIdx.x * blockDim.x) + threadIdx.x); // block index
	if(bi < n) {
		int p = bi*4;
		uint32_t s0,s1,s2,s3,t0,t1,t2,t3;
		uint8_t* s0p = (uint8_t*) &s0;
		uint8_t* s1p = (uint8_t*) &s1;
		uint8_t* s2p = (uint8_t*) &s2;
		uint8_t* s3p = (uint8_t*) &s3;
		uint8_t* t0p = (uint8_t*) &t0;
		uint8_t* t1p = (uint8_t*) &t1;
		uint8_t* t2p = (uint8_t*) &t2;
		uint8_t* t3p = (uint8_t*) &t3;
		int key_index_sum = 0;

		/*
		 * map byte array block to cipher state
		 * and add initial round key:
		 */
		__LOG_TRACE__("p %d: d[0] => 0x%04x",p,d[p]);
		__LOG_TRACE__("p %d: d[1] => 0x%04x",p,d[p+1]);
		__LOG_TRACE__("p %d: d[2] => 0x%04x",p,d[p+2]);
		__LOG_TRACE__("p %d: d[3] => 0x%04x",p,d[p+3]);
		__LOG_TRACE__("p %d: k[0] => 0x%04x",p,k[0]);
		__LOG_TRACE__("p %d: k[1] => 0x%04x",p,k[1]);
		__LOG_TRACE__("p %d: k[2] => 0x%04x",p,k[2]);
		__LOG_TRACE__("p %d: k[3] => 0x%04x",p,k[3]);
		s0 = d[p]   ^ k[0];
		s1 = d[p+1] ^ k[1];
		s2 = d[p+2] ^ k[2];
		s3 = d[p+3] ^ k[3];
		__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0);
		__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1);
		__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2);
		__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3);

		// 8 rounds - in each loop we do two rounds
		#pragma unroll
		for(int r2 = 1; r2 <= 4; r2++) {
			__LOG_TRACE__("p %d: s0p[0] => 0x%04x",p,s0p[0]);
			__LOG_TRACE__("p %d: s1p[1] => 0x%04x",p,s1p[1]);
			__LOG_TRACE__("p %d: s2p[2] => 0x%04x",p,s2p[2]);
			__LOG_TRACE__("p %d: s3p[3] => 0x%04x",p,(s3 >> 24));
			__LOG_TRACE__("p %d: T0[s0p[0]] => 0x%04x",p,T0[s0p[0]]);
			__LOG_TRACE__("p %d: T1[s1p[1]] => 0x%04x",p,T1[s1p[1]]);
			__LOG_TRACE__("p %d: T2[s2p[2]] => 0x%04x",p,T2[s2p[2]]);
			__LOG_TRACE__("p %d: T3[s3p[3]] => 0x%04x",p,T3[s3p[3]]);
			__LOG_TRACE__("p %d: k[%d] => 0x%04x",p,(r2*8)-4 , k[(r2*8)-4]);
			t0 = T0[s0p[0]] ^ T1[s1p[1]] ^ T2[s2p[2]] ^ T3[s3p[3]] ^ k[(r2*8)-4];
			t1 = T0[s1p[0]] ^ T1[s2p[1]] ^ T2[s3p[2]] ^ T3[s0p[3]] ^ k[(r2*8)-3];
			t2 = T0[s2p[0]] ^ T1[s3p[1]] ^ T2[s0p[2]] ^ T3[s1p[3]] ^ k[(r2*8)-2];
			t3 = T0[s3p[0]] ^ T1[s0p[1]] ^ T2[s1p[2]] ^ T3[s2p[3]] ^ k[(r2*8)-1];
			__LOG_TRACE__("p %d: t0 => 0x%04x",p,t0);
			__LOG_TRACE__("p %d: t1 => 0x%04x",p,t1);
			__LOG_TRACE__("p %d: t2 => 0x%04x",p,t2);
			__LOG_TRACE__("p %d: t3 => 0x%04x",p,t3);

			s0 = T0[t0p[0]] ^ T1[t1p[1]] ^ T2[t2p[2]] ^ T3[t3p[3]] ^ k[(r2*8)  ];
			s1 = T0[t1p[0]] ^ T1[t2p[1]] ^ T2[t3p[2]] ^ T3[t0p[3]] ^ k[(r2*8)+1];
			s2 = T0[t2p[0]] ^ T1[t3p[1]] ^ T2[t0p[2]] ^ T3[t1p[3]] ^ k[(r2*8)+2];
			s3 = T0[t3p[0]] ^ T1[t0p[1]] ^ T2[t1p[2]] ^ T3[t2p[3]] ^ k[(r2*8)+3];
			__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0);
			__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1);
			__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2);
			__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3);
		}

		if(key_bits >= 192) {
			key_index_sum = 8;

			t0 = T0[s0p[0]] ^ T1[s1p[1]] ^ T2[s2p[2]] ^ T3[s3p[3]] ^ k[36];
			t1 = T0[s1p[0]] ^ T1[s2p[1]] ^ T2[s3p[2]] ^ T3[s0p[3]] ^ k[37];
			t2 = T0[s2p[0]] ^ T1[s3p[1]] ^ T2[s0p[2]] ^ T3[s1p[3]] ^ k[38];
			t3 = T0[s3p[0]] ^ T1[s0p[1]] ^ T2[s1p[2]] ^ T3[s2p[3]] ^ k[39];
			__LOG_TRACE__("p %d: t0 => 0x%04x",p,t0);
			__LOG_TRACE__("p %d: t1 => 0x%04x",p,t1);
			__LOG_TRACE__("p %d: t2 => 0x%04x",p,t2);
			__LOG_TRACE__("p %d: t3 => 0x%04x",p,t3);

			s0 = T0[t0p[0]] ^ T1[t1p[1]] ^ T2[t2p[2]] ^ T3[t3p[3]] ^ k[40];
			s1 = T0[t1p[0]] ^ T1[t2p[1]] ^ T2[t3p[2]] ^ T3[t0p[3]] ^ k[41];
			s2 = T0[t2p[0]] ^ T1[t3p[1]] ^ T2[t0p[2]] ^ T3[t1p[3]] ^ k[42];
			s3 = T0[t3p[0]] ^ T1[t0p[1]] ^ T2[t1p[2]] ^ T3[t2p[3]] ^ k[43];
			__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0);
			__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1);
			__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2);
			__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3);

			if(key_bits == 256) {
				key_index_sum = 16;

				t0 = T0[s0p[0]] ^ T1[s1p[1]] ^ T2[s2p[2]] ^ T3[s3p[3]] ^ k[44];
				t1 = T0[s1p[0]] ^ T1[s2p[1]] ^ T2[s3p[2]] ^ T3[s0p[3]] ^ k[45];
				t2 = T0[s2p[0]] ^ T1[s3p[1]] ^ T2[s0p[2]] ^ T3[s1p[3]] ^ k[46];
				t3 = T0[s3p[0]] ^ T1[s0p[1]] ^ T2[s1p[2]] ^ T3[s2p[3]] ^ k[47];
				__LOG_TRACE__("p %d: t0 => 0x%04x",p,t0);
				__LOG_TRACE__("p %d: t1 => 0x%04x",p,t1);
				__LOG_TRACE__("p %d: t2 => 0x%04x",p,t2);
				__LOG_TRACE__("p %d: t3 => 0x%04x",p,t3);

				s0 = T0[t0p[0]] ^ T1[t1p[1]] ^ T2[t2p[2]] ^ T3[t3p[3]] ^ k[48];
				s1 = T0[t1p[0]] ^ T1[t2p[1]] ^ T2[t3p[2]] ^ T3[t0p[3]] ^ k[49];
				s2 = T0[t2p[0]] ^ T1[t3p[1]] ^ T2[t0p[2]] ^ T3[t1p[3]] ^ k[50];
				s3 = T0[t3p[0]] ^ T1[t0p[1]] ^ T2[t1p[2]] ^ T3[t2p[3]] ^ k[51];
				__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0);
				__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1);
				__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2);
				__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3);
			}
		}

		t0 = T0[s0p[0]] ^ T1[s1p[1]] ^ T2[s2p[2]] ^ T3[s3p[3]] ^ k[36+key_index_sum];
		t1 = T0[s1p[0]] ^ T1[s2p[1]] ^ T2[s3p[2]] ^ T3[s0p[3]] ^ k[37+key_index_sum];
		t2 = T0[s2p[0]] ^ T1[s3p[1]] ^ T2[s0p[2]] ^ T3[s1p[3]] ^ k[38+key_index_sum];
		t3 = T0[s3p[0]] ^ T1[s0p[1]] ^ T2[s1p[2]] ^ T3[s2p[3]] ^ k[39+key_index_sum];
		__LOG_TRACE__("p %d: t0 => 0x%04x",p,t0);
		__LOG_TRACE__("p %d: t1 => 0x%04x",p,t1);
		__LOG_TRACE__("p %d: t2 => 0x%04x",p,t2);
		__LOG_TRACE__("p %d: t3 => 0x%04x",p,t3);

		// last round - save result
		s0 =
			(T2[t0p[0]] & 0x000000ff) ^
			(T3[t1p[1]] & 0x0000ff00) ^
			(T0[t2p[2]] & 0x00ff0000) ^
			(T1[t3p[3]] & 0xff000000) ^
			k[40+key_index_sum];
		__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0);
		d[p] = s0;
		s1 =
			(T2[t1p[0]] & 0x000000ff) ^
			(T3[t2p[1]] & 0x0000ff00) ^
			(T0[t3p[2]] & 0x00ff0000) ^
			(T1[t0p[3]] & 0xff000000) ^
			k[41+key_index_sum];
		__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1);
		d[p+1] = s1;
		s2 =
			(T2[t2p[0]] & 0x000000ff) ^
			(T3[t3p[1]] & 0x0000ff00) ^
			(T0[t0p[2]] & 0x00ff0000) ^
			(T1[t1p[3]] & 0xff000000) ^
			k[42+key_index_sum];
		__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2);
		d[p+2] = s2;
		s3 =
			(T2[t3p[0]] & 0x000000ff) ^
			(T3[t0p[1]] & 0x0000ff00) ^
			(T0[t1p[2]] & 0x00ff0000) ^
			(T2[t2p[3]] & 0xff000000) ^
			k[43+key_index_sum];
		__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3);
		d[p+3] = s3;
	}
}

__global__ void __cuda_ecb_aes_16b_ptr_decrypt__(
		  int n,
		  uint32_t* d,
	  	  uint32_t* k,
	  	  int key_bits,
	  	  uint32_t* T0,
	  	  uint32_t* T1,
	  	  uint32_t* T2,
	  	  uint32_t* T3,
	  	  uint8_t* T4
    )
{
	int bi = ((blockIdx.x * blockDim.x) + threadIdx.x); // block index
	if(bi < n) {
		int p = bi*4;
		uint32_t s0,s1,s2,s3,t0,t1,t2,t3;
		uint8_t* s0p = (uint8_t*) &s0;
		uint8_t* s1p = (uint8_t*) &s1;
		uint8_t* s2p = (uint8_t*) &s2;
		uint8_t* s3p = (uint8_t*) &s3;
		uint8_t* t0p = (uint8_t*) &t0;
		uint8_t* t1p = (uint8_t*) &t1;
		uint8_t* t2p = (uint8_t*) &t2;
		uint8_t* t3p = (uint8_t*) &t3;
		int key_index_sum = 0;

		/*
		 * map byte array block to cipher state
		 * and add initial round key:
		 */
		__LOG_TRACE__("p %d: d[0] => 0x%04x",p,d[p]);
		__LOG_TRACE__("p %d: d[1] => 0x%04x",p,d[p+1]);
		__LOG_TRACE__("p %d: d[2] => 0x%04x",p,d[p+2]);
		__LOG_TRACE__("p %d: d[3] => 0x%04x",p,d[p+3]);
		__LOG_TRACE__("p %d: k[0] => 0x%04x",p,k[0]);
		__LOG_TRACE__("p %d: k[1] => 0x%04x",p,k[1]);
		__LOG_TRACE__("p %d: k[2] => 0x%04x",p,k[2]);
		__LOG_TRACE__("p %d: k[3] => 0x%04x",p,k[3]);
		s0 = d[p]   ^ k[0];
		s1 = d[p+1] ^ k[1];
		s2 = d[p+2] ^ k[2];
		s3 = d[p+3] ^ k[3];
		__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0);
		__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1);
		__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2);
		__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3);

		// 8 rounds - in each loop we do two rounds
		#pragma unroll
		for(int r2 = 1; r2 <= 4; r2++) {
			__LOG_TRACE__("p %d: s0p[0] => 0x%04x",p,s0p[0]);
			__LOG_TRACE__("p %d: s1p[1] => 0x%04x",p,s1p[1]);
			__LOG_TRACE__("p %d: s2p[2] => 0x%04x",p,s2p[2]);
			__LOG_TRACE__("p %d: s3p[3] => 0x%04x",p,(s3 >> 24));
			__LOG_TRACE__("p %d: T0[s0p[0]] => 0x%04x",p,T0[s0p[0]]);
			__LOG_TRACE__("p %d: T1[s1p[1]] => 0x%04x",p,T1[s1p[1]]);
			__LOG_TRACE__("p %d: T2[s2p[2]] => 0x%04x",p,T2[s2p[2]]);
			__LOG_TRACE__("p %d: T3[s3p[3]] => 0x%04x",p,T3[s3p[3]]);
			__LOG_TRACE__("p %d: k[%d] => 0x%04x",p,(r2*8)-4 , k[(r2*8)-4]);
			t0 = T0[s0p[0]] ^ T1[s3p[1]] ^ T2[s2p[2]] ^ T3[s1p[3]] ^ k[(r2*8)-4];
			t1 = T0[s1p[0]] ^ T1[s0p[1]] ^ T2[s3p[2]] ^ T3[s2p[3]] ^ k[(r2*8)-3];
			t2 = T0[s2p[0]] ^ T1[s1p[1]] ^ T2[s0p[2]] ^ T3[s3p[3]] ^ k[(r2*8)-2];
			t3 = T0[s3p[0]] ^ T1[s2p[1]] ^ T2[s1p[2]] ^ T3[s0p[3]] ^ k[(r2*8)-1];
			__LOG_TRACE__("p %d: t0 => 0x%04x",p,t0);
			__LOG_TRACE__("p %d: t1 => 0x%04x",p,t1);
			__LOG_TRACE__("p %d: t2 => 0x%04x",p,t2);
			__LOG_TRACE__("p %d: t3 => 0x%04x",p,t3);

			s0 = T0[t0p[0]] ^ T1[t3p[1]] ^ T2[t2p[2]] ^ T3[t1p[3]] ^ k[(r2*8)  ];
			s1 = T0[t1p[0]] ^ T1[t0p[1]] ^ T2[t3p[2]] ^ T3[t2p[3]] ^ k[(r2*8)+1];
			s2 = T0[t2p[0]] ^ T1[t1p[1]] ^ T2[t0p[2]] ^ T3[t3p[3]] ^ k[(r2*8)+2];
			s3 = T0[t3p[0]] ^ T1[t2p[1]] ^ T2[t1p[2]] ^ T3[t0p[3]] ^ k[(r2*8)+3];
			__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0);
			__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1);
			__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2);
			__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3);
		}

		if(key_bits >= 192) {
			key_index_sum = 8;

			t0 = T0[s0p[0]] ^ T1[s3p[1]] ^ T2[s2p[2]] ^ T3[s1p[3]] ^ k[36];
			t1 = T0[s1p[0]] ^ T1[s0p[1]] ^ T2[s3p[2]] ^ T3[s2p[3]] ^ k[37];
			t2 = T0[s2p[0]] ^ T1[s1p[1]] ^ T2[s0p[2]] ^ T3[s3p[3]] ^ k[38];
			t3 = T0[s3p[0]] ^ T1[s2p[1]] ^ T2[s1p[2]] ^ T3[s0p[3]] ^ k[39];
			__LOG_TRACE__("p %d: t0 => 0x%04x",p,t0);
			__LOG_TRACE__("p %d: t1 => 0x%04x",p,t1);
			__LOG_TRACE__("p %d: t2 => 0x%04x",p,t2);
			__LOG_TRACE__("p %d: t3 => 0x%04x",p,t3);

			s0 = T0[t0p[0]] ^ T1[t3p[1]] ^ T2[t2p[2]] ^ T3[t1p[3]] ^ k[40];
			s1 = T0[t1p[0]] ^ T1[t0p[1]] ^ T2[t3p[2]] ^ T3[t2p[3]] ^ k[41];
			s2 = T0[t2p[0]] ^ T1[t1p[1]] ^ T2[t0p[2]] ^ T3[t3p[3]] ^ k[42];
			s3 = T0[t3p[0]] ^ T1[t2p[1]] ^ T2[t1p[2]] ^ T3[t0p[3]] ^ k[43];
			__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0);
			__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1);
			__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2);
			__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3);

			if(key_bits == 256) {
				key_index_sum = 16;

				t0 = T0[s0p[0]] ^ T1[s3p[1]] ^ T2[s2p[2]] ^ T3[s1p[3]] ^ k[44];
				t1 = T0[s1p[0]] ^ T1[s0p[1]] ^ T2[s3p[2]] ^ T3[s2p[3]] ^ k[45];
				t2 = T0[s2p[0]] ^ T1[s1p[1]] ^ T2[s0p[2]] ^ T3[s3p[3]] ^ k[46];
				t3 = T0[s3p[0]] ^ T1[s2p[1]] ^ T2[s1p[2]] ^ T3[s0p[3]] ^ k[47];
				__LOG_TRACE__("p %d: t0 => 0x%04x",p,t0);
				__LOG_TRACE__("p %d: t1 => 0x%04x",p,t1);
				__LOG_TRACE__("p %d: t2 => 0x%04x",p,t2);
				__LOG_TRACE__("p %d: t3 => 0x%04x",p,t3);

				s0 = T0[t0p[0]] ^ T1[t3p[1]] ^ T2[t2p[2]] ^ T3[t1p[3]] ^ k[48];
				s1 = T0[t1p[0]] ^ T1[t0p[1]] ^ T2[t3p[2]] ^ T3[t2p[3]] ^ k[49];
				s2 = T0[t2p[0]] ^ T1[t1p[1]] ^ T2[t0p[2]] ^ T3[t3p[3]] ^ k[50];
				s3 = T0[t3p[0]] ^ T1[t2p[1]] ^ T2[t1p[2]] ^ T3[t0p[3]] ^ k[51];
				__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0);
				__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1);
				__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2);
				__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3);

			}
		}

		t0 = T0[s0p[0]] ^ T1[s3p[1]] ^ T2[s2p[2]] ^ T3[s1p[3]] ^ k[36+key_index_sum];
		t1 = T0[s1p[0]] ^ T1[s0p[1]] ^ T2[s3p[2]] ^ T3[s2p[3]] ^ k[37+key_index_sum];
		t2 = T0[s2p[0]] ^ T1[s1p[1]] ^ T2[s0p[2]] ^ T3[s3p[3]] ^ k[38+key_index_sum];
		t3 = T0[s3p[0]] ^ T1[s2p[1]] ^ T2[s1p[2]] ^ T3[s0p[3]] ^ k[39+key_index_sum];
		__LOG_TRACE__("p %d: t0 => 0x%04x",p,t0);
		__LOG_TRACE__("p %d: t1 => 0x%04x",p,t1);
		__LOG_TRACE__("p %d: t2 => 0x%04x",p,t2);
		__LOG_TRACE__("p %d: t3 => 0x%04x",p,t3);

		// last round - save result
		s0 =
			((uint32_t)T4[t0p[0]]      ) ^
			((uint32_t)T4[t3p[1]] <<  8) ^
			((uint32_t)T4[t2p[2]] << 16) ^
			((uint32_t)T4[t1p[3]] << 24) ^
			k[40+key_index_sum];
		__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0);
		d[p] = s0;
		s1 =
			((uint32_t)T4[t1p[0]]      ) ^
			((uint32_t)T4[t0p[1]] <<  8) ^
			((uint32_t)T4[t3p[2]] << 16) ^
			((uint32_t)T4[t2p[3]] << 24) ^
			k[41+key_index_sum];
		__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1);
		d[p+1] = s1;
		s2 =
			((uint32_t)T4[t2p[0]]      ) ^
			((uint32_t)T4[t1p[1]] <<  8) ^
			((uint32_t)T4[t0p[2]] << 16) ^
			((uint32_t)T4[t3p[3]] << 24) ^
			k[42+key_index_sum];
		__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2);
		d[p+2] = s2;
		s3 =
			((uint32_t)T4[t3p[0]]      ) ^
			((uint32_t)T4[t2p[1]] <<  8) ^
			((uint32_t)T4[t1p[2]] << 16) ^
			((uint32_t)T4[t0p[3]] << 24) ^
			k[43+key_index_sum];
		__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3);
		d[p+3] = s3;
	}
}

void cuda_ecb_aes_16b_ptr_encrypt(
		  	  int gridSize,
		  	  int threadsPerBlock,
		  	  int n_blocks,
		  	  unsigned char data[],
		  	  uint32_t* expanded_key,
		  	  int key_bits,
		  	  uint32_t* deviceTe0,
		  	  uint32_t* deviceTe1,
		  	  uint32_t* deviceTe2,
		  	  uint32_t* deviceTe3
	      )
{
	__cuda_ecb_aes_16b_ptr_encrypt__<<<gridSize,threadsPerBlock>>>(
			n_blocks,
			(uint32_t*)data,
			expanded_key,
		  	 key_bits,
	   		deviceTe0,
	   		deviceTe1,
	   		deviceTe2,
	   		deviceTe3
	);
}

void cuda_ecb_aes_16b_ptr_decrypt(
		  	  int gridSize,
		  	  int threadsPerBlock,
		  	  int n_blocks,
		  	  unsigned char data[],
		  	  uint32_t* expanded_key,
		  	  int key_bits,
		  	  uint32_t* deviceTd0,
		  	  uint32_t* deviceTd1,
		  	  uint32_t* deviceTd2,
		  	  uint32_t* deviceTd3,
		  	  uint8_t* deviceTd4
	      )
{
	__cuda_ecb_aes_16b_ptr_decrypt__<<<gridSize,threadsPerBlock>>>(
			n_blocks,
			(uint32_t*)data,
			expanded_key,
		  	 key_bits,
	   		deviceTd0,
	   		deviceTd1,
	   		deviceTd2,
	   		deviceTd3,
	   		deviceTd4
	);
}
