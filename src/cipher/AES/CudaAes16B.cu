#include "hip/hip_runtime.h"
/*
 *  Copyright (C) 2017 Jesus Martin Berlanga. All Rights Reserved.
 *
 *  This file is part of Paracrypt.
 *
 *  Paracrypt is free software: you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation, either version 3 of the License, or
 *  (at your option) any later version.
 *
 *  Paracrypt is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with Paracrypt.  If not, see <http://www.gnu.org/licenses/>.
 *
 */

#include "cipher/BlockCipher.hpp"
#include "CudaAes16B.cuh"
#include "cuda_logging.cuh"

__global__ void __cuda_ecb_aes_16b_encrypt__(
		const paracrypt::BlockCipher::Mode m,
		unsigned int n,
		unsigned int offset,
		const uint32_t* d,
		uint32_t* out,
		uint32_t* neigh,
		uint32_t* iv,
		uint32_t* k,
		const int key_bits,
		uint32_t* T0,
		uint32_t* T1,
		uint32_t* T2,
		uint32_t* T3
    )
{
	unsigned int bi = ((blockIdx.x * blockDim.x) + threadIdx.x); // block index
	if(bi < n) {
		unsigned int p = bi*4;
		uint32_t s0,s1,s2,s3,t0,t1,t2,t3;
		unsigned int key_index_sum = 0;

		/*
		 * map byte array block to cipher state
		 * and add initial round key:
		 */
		__LOG_TRACE__("p %d: d[0] => 0x%04x",p,d[p]);
		__LOG_TRACE__("p %d: d[1] => 0x%04x",p,d[p+1]);
		__LOG_TRACE__("p %d: d[2] => 0x%04x",p,d[p+2]);
		__LOG_TRACE__("p %d: d[3] => 0x%04x",p,d[p+3]);
		__LOG_TRACE__("p %d: k[0] => 0x%04x",p,k[0]);
		__LOG_TRACE__("p %d: k[1] => 0x%04x",p,k[1]);
		__LOG_TRACE__("p %d: k[2] => 0x%04x",p,k[2]);
		__LOG_TRACE__("p %d: k[3] => 0x%04x",p,k[3]);
		s0 = d[p]   ^ k[0];
		s1 = d[p+1] ^ k[1];
		s2 = d[p+2] ^ k[2];
		s3 = d[p+3] ^ k[3];
		__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0);
		__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1);
		__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2);
		__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3);

		// 8 rounds - in each loop we do two rounds
		#pragma unroll
		for(int r2 = 1; r2 <= 4; r2++) {
			__LOG_TRACE__("p %d: (s0      ) & 0xff => 0x%04x",p,(s0      ) & 0xff);
			__LOG_TRACE__("p %d: (s1 >>  8) & 0xff => 0x%04x",p,(s1 >>  8) & 0xff);
			__LOG_TRACE__("p %d: (s2 >> 16) & 0xff => 0x%04x",p,(s2 >> 16) & 0xff);
			__LOG_TRACE__("p %d: (s3 >> 24)        => 0x%04x",p,(s3 >> 24));
			__LOG_TRACE__("p %d: T0[(s0      ) & 0xff] => 0x%04x",p,T0[(s0      ) & 0xff]);
			__LOG_TRACE__("p %d: T1[(s1 >>  8) & 0xff] => 0x%04x",p,T1[(s1 >>  8) & 0xff]);
			__LOG_TRACE__("p %d: T2[(s2 >> 16) & 0xff] => 0x%04x",p,T2[(s2 >> 16) & 0xff]);
			__LOG_TRACE__("p %d: T3[(s3 >> 24)       ] => 0x%04x",p,T3[(s3 >> 24)       ]);
			__LOG_TRACE__("p %d: k[%d] => 0x%04x",p,(r2*8)-4 , k[(r2*8)-4]);
			t0 =
				T0[(s0      ) & 0xff] ^
				T1[(s1 >>  8) & 0xff] ^
				T2[(s2 >> 16) & 0xff] ^
				T3[(s3 >> 24)       ] ^
				k[(r2*8)-4];
			t1 =
				T0[(s1      ) & 0xff] ^
				T1[(s2 >>  8) & 0xff] ^
				T2[(s3 >> 16) & 0xff] ^
				T3[(s0 >> 24)       ] ^
				k[(r2*8)-3];
			t2 =
				T0[(s2      ) & 0xff] ^
				T1[(s3 >>  8) & 0xff] ^
				T2[(s0 >> 16) & 0xff] ^
				T3[(s1 >> 24)       ] ^
				k[(r2*8)-2];
			t3 =
				T0[(s3      ) & 0xff] ^
				T1[(s0 >>  8) & 0xff] ^
				T2[(s1 >> 16) & 0xff] ^
				T3[(s2 >> 24)       ] ^
				k[(r2*8)-1];
			__LOG_TRACE__("p %d: t0 => 0x%04x",p,t0);
			__LOG_TRACE__("p %d: t1 => 0x%04x",p,t1);
			__LOG_TRACE__("p %d: t2 => 0x%04x",p,t2);
			__LOG_TRACE__("p %d: t3 => 0x%04x",p,t3);

			s0 =
				T0[(t0      ) & 0xff] ^
				T1[(t1 >>  8) & 0xff] ^
				T2[(t2 >> 16) & 0xff] ^
				T3[(t3 >> 24)       ] ^
				k[(r2*8)  ];
			s1 =
				T0[(t1      ) & 0xff] ^
				T1[(t2 >>  8) & 0xff] ^
				T2[(t3 >> 16) & 0xff] ^
				T3[(t0 >> 24)       ] ^
				k[(r2*8)+1];
			s2 =
				T0[(t2      ) & 0xff] ^
				T1[(t3 >>  8) & 0xff] ^
				T2[(t0 >> 16) & 0xff] ^
				T3[(t1 >> 24)       ] ^
				k[(r2*8)+2];
			s3 =
				T0[(t3      ) & 0xff] ^
				T1[(t0 >>  8) & 0xff] ^
				T2[(t1 >> 16) & 0xff] ^
				T3[(t2 >> 24)       ] ^
				k[(r2*8)+3];
			__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0);
			__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1);
			__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2);
			__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3);
		}

		if(key_bits >= 192) {
			key_index_sum = 8;

			t0 =
				T0[(s0      ) & 0xff] ^
				T1[(s1 >>  8) & 0xff] ^
				T2[(s2 >> 16) & 0xff] ^
				T3[(s3 >> 24)       ] ^
				k[36];
			t1 =
				T0[(s1      ) & 0xff] ^
				T1[(s2 >>  8) & 0xff] ^
				T2[(s3 >> 16) & 0xff] ^
				T3[(s0 >> 24)       ] ^
				k[37];
			t2 =
				T0[(s2      ) & 0xff] ^
				T1[(s3 >>  8) & 0xff] ^
				T2[(s0 >> 16) & 0xff] ^
				T3[(s1 >> 24)       ] ^
				k[38];
			t3 =
				T0[(s3      ) & 0xff] ^
				T1[(s0 >>  8) & 0xff] ^
				T2[(s1 >> 16) & 0xff] ^
				T3[(s2 >> 24)       ] ^
				k[39];
			__LOG_TRACE__("p %d: t0 => 0x%04x",p,t0);
			__LOG_TRACE__("p %d: t1 => 0x%04x",p,t1);
			__LOG_TRACE__("p %d: t2 => 0x%04x",p,t2);
			__LOG_TRACE__("p %d: t3 => 0x%04x",p,t3);

			s0 =
				T0[(t0      ) & 0xff] ^
				T1[(t1 >>  8) & 0xff] ^
				T2[(t2 >> 16) & 0xff] ^
				T3[(t3 >> 24)       ] ^
				k[40];
			s1 =
				T0[(t1      ) & 0xff] ^
				T1[(t2 >>  8) & 0xff] ^
				T2[(t3 >> 16) & 0xff] ^
				T3[(t0 >> 24)       ] ^
				k[41];
			s2 =
				T0[(t2      ) & 0xff] ^
				T1[(t3 >>  8) & 0xff] ^
				T2[(t0 >> 16) & 0xff] ^
				T3[(t1 >> 24)       ] ^
				k[42];
			s3 =
				T0[(t3      ) & 0xff] ^
				T1[(t0 >>  8) & 0xff] ^
				T2[(t1 >> 16) & 0xff] ^
				T3[(t2 >> 24)       ] ^
				k[43];
			__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0);
			__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1);
			__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2);
			__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3);

			if(key_bits == 256) {
				key_index_sum = 16;

				t0 =
					T0[(s0      ) & 0xff] ^
					T1[(s1 >>  8) & 0xff] ^
					T2[(s2 >> 16) & 0xff] ^
					T3[(s3 >> 24)       ] ^
					k[44];
				t1 =
					T0[(s1      ) & 0xff] ^
					T1[(s2 >>  8) & 0xff] ^
					T2[(s3 >> 16) & 0xff] ^
					T3[(s0 >> 24)       ] ^
					k[45];
				t2 =
					T0[(s2      ) & 0xff] ^
					T1[(s3 >>  8) & 0xff] ^
					T2[(s0 >> 16) & 0xff] ^
					T3[(s1 >> 24)       ] ^
					k[46];
				t3 =
					T0[(s3      ) & 0xff] ^
					T1[(s0 >>  8) & 0xff] ^
					T2[(s1 >> 16) & 0xff] ^
					T3[(s2 >> 24)       ] ^
					k[47];
				__LOG_TRACE__("p %d: t0 => 0x%04x",p,t0);
				__LOG_TRACE__("p %d: t1 => 0x%04x",p,t1);
				__LOG_TRACE__("p %d: t2 => 0x%04x",p,t2);
				__LOG_TRACE__("p %d: t3 => 0x%04x",p,t3);

				s0 =
					T0[(t0      ) & 0xff] ^
					T1[(t1 >>  8) & 0xff] ^
					T2[(t2 >> 16) & 0xff] ^
					T3[(t3 >> 24)       ] ^
					k[48];
				s1 =
					T0[(t1      ) & 0xff] ^
					T1[(t2 >>  8) & 0xff] ^
					T2[(t3 >> 16) & 0xff] ^
					T3[(t0 >> 24)       ] ^
					k[49];
				s2 =
					T0[(t2      ) & 0xff] ^
					T1[(t3 >>  8) & 0xff] ^
					T2[(t0 >> 16) & 0xff] ^
					T3[(t1 >> 24)       ] ^
					k[50];
				s3 =
					T0[(t3      ) & 0xff] ^
					T1[(t0 >>  8) & 0xff] ^
					T2[(t1 >> 16) & 0xff] ^
					T3[(t2 >> 24)       ] ^
					k[51];
				__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0);
				__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1);
				__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2);
				__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3);
			}
		}

		t0 =
			T0[(s0      ) & 0xff] ^
			T1[(s1 >>  8) & 0xff] ^
			T2[(s2 >> 16) & 0xff] ^
			T3[(s3 >> 24)       ] ^
			k[36+key_index_sum];
		t1 =
			T0[(s1      ) & 0xff] ^
			T1[(s2 >>  8) & 0xff] ^
			T2[(s3 >> 16) & 0xff] ^
			T3[(s0 >> 24)       ] ^
			k[37+key_index_sum];
		t2 =
			T0[(s2      ) & 0xff] ^
			T1[(s3 >>  8) & 0xff] ^
			T2[(s0 >> 16) & 0xff] ^
			T3[(s1 >> 24)       ] ^
			k[38+key_index_sum];
		t3 =
			T0[(s3      ) & 0xff] ^
			T1[(s0 >>  8) & 0xff] ^
			T2[(s1 >> 16) & 0xff] ^
			T3[(s2 >> 24)       ] ^
			k[39+key_index_sum];
		__LOG_TRACE__("p %d: t0 => 0x%04x",p,t0);
		__LOG_TRACE__("p %d: t1 => 0x%04x",p,t1);
		__LOG_TRACE__("p %d: t2 => 0x%04x",p,t2);
		__LOG_TRACE__("p %d: t3 => 0x%04x",p,t3);

		// last round - save result
		s0 =
			(T2[(t0      ) & 0xff] & 0x000000ff) ^
			(T3[(t1 >>  8) & 0xff] & 0x0000ff00) ^
			(T0[(t2 >> 16) & 0xff] & 0x00ff0000) ^
			(T1[(t3 >> 24)       ] & 0xff000000) ^
			k[40+key_index_sum];
		__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0);
		out[p] = s0;
		s1 =
			(T2[(t1      ) & 0xff] & 0x000000ff) ^
			(T3[(t2 >>  8) & 0xff] & 0x0000ff00) ^
			(T0[(t3 >> 16) & 0xff] & 0x00ff0000) ^
			(T1[(t0 >> 24)       ] & 0xff000000) ^
			k[41+key_index_sum];
		__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1);
		out[p+1] = s1;
		s2 =
			(T2[(t2      ) & 0xff] & 0x000000ff) ^
			(T3[(t3 >>  8) & 0xff] & 0x0000ff00) ^
			(T0[(t0 >> 16) & 0xff] & 0x00ff0000) ^
			(T1[(t1 >> 24)       ] & 0xff000000) ^
			k[42+key_index_sum];
		__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2);
		out[p+2] = s2;
		s3 =
			(T2[(t3      ) & 0xff] & 0x000000ff) ^
			(T3[(t0 >>  8) & 0xff] & 0x0000ff00) ^
			(T0[(t1 >> 16) & 0xff] & 0x00ff0000) ^
			(T2[(t2 >> 24)       ] & 0xff000000) ^
			k[43+key_index_sum];
		__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3);
		out[p+3] = s3;
	}
}

__global__ void __cuda_ecb_aes_16b_decrypt__(
        const paracrypt::BlockCipher::Mode m,
		unsigned int n,
		unsigned int offset,
		const uint32_t* d,
		uint32_t* out,
		uint32_t* neigh,
		uint32_t* iv,
		uint32_t* k,
		const int key_bits,
		uint32_t* T0,
		uint32_t* T1,
		uint32_t* T2,
		uint32_t* T3,
		uint8_t* T4
    )
{
	unsigned int bi = ((blockIdx.x * blockDim.x) + threadIdx.x); // block index
	unsigned int p;
	uint32_t s0,s1,s2,s3,t0,t1,t2,t3;
	if(bi < n) {
		p = bi*4;
		unsigned int key_index_sum = 0;

		/*
		 * map byte array block to cipher state
		 * and add initial round key:
		 */
		if(m == paracrypt::BlockCipher::CFB || m == paracrypt::BlockCipher::GCM) {
			// - The counter is the block index: offset+bi
			// - For security the counter is combined with a noence: global_bi ^ iv
            // - Initial round: ^k
			unsigned int global_bi = offset+bi;
			s0 = global_bi ^ iv[0] ^ k[0];
			s1 = global_bi ^ iv[1] ^ k[1];
			s2 = global_bi ^ iv[2] ^ k[2];
			s3 = global_bi ^ iv[3] ^ k[3];
		}
		else {
			__LOG_TRACE__("p %d: d[0] => 0x%04x",p,d[p]);
			__LOG_TRACE__("p %d: d[1] => 0x%04x",p,d[p+1]);
			__LOG_TRACE__("p %d: d[2] => 0x%04x",p,d[p+2]);
			__LOG_TRACE__("p %d: d[3] => 0x%04x",p,d[p+3]);
			__LOG_TRACE__("p %d: k[0] => 0x%04x",p,k[0]);
			__LOG_TRACE__("p %d: k[1] => 0x%04x",p,k[1]);
			__LOG_TRACE__("p %d: k[2] => 0x%04x",p,k[2]);
			__LOG_TRACE__("p %d: k[3] => 0x%04x",p,k[3]);
			s0 = d[p]   ^ k[0];
			s1 = d[p+1] ^ k[1];
			s2 = d[p+2] ^ k[2];
			s3 = d[p+3] ^ k[3];
		}
		__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0);
		__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1);
		__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2);
		__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3);

		// 8 rounds - in each loop we do two rounds
		#pragma unroll
		for(int r2 = 1; r2 <= 4; r2++) {
			__LOG_TRACE__("p %d: (s0      ) & 0xff => 0x%04x",p,(s0      ) & 0xff);
			__LOG_TRACE__("p %d: (s3 >>  8) & 0xff => 0x%04x",p,(s3 >>  8) & 0xff);
			__LOG_TRACE__("p %d: (s2 >> 16) & 0xff => 0x%04x",p,(s2 >> 16) & 0xff);
			__LOG_TRACE__("p %d: (s1 >> 24)        => 0x%04x",p,(s1 >> 24));
			__LOG_TRACE__("p %d: T0[(s0      ) & 0xff] => 0x%04x",p,T0[(s0      ) & 0xff]);
			__LOG_TRACE__("p %d: T1[(s3 >>  8) & 0xff] => 0x%04x",p,T1[(s3 >>  8) & 0xff]);
			__LOG_TRACE__("p %d: T2[(s2 >> 16) & 0xff] => 0x%04x",p,T2[(s2 >> 16) & 0xff]);
			__LOG_TRACE__("p %d: T3[(s1 >> 24)       ] => 0x%04x",p,T3[(s1 >> 24)       ]);
			__LOG_TRACE__("p %d: k[%d] => 0x%04x",p,(r2*8)-4 , k[(r2*8)-4]);
			t0 =
				T0[(s0      ) & 0xff] ^
				T1[(s3 >>  8) & 0xff] ^
				T2[(s2 >> 16) & 0xff] ^
				T3[(s1 >> 24)       ] ^
				k[(r2*8)-4];
			t1 =
				T0[(s1      ) & 0xff] ^
				T1[(s0 >>  8) & 0xff] ^
				T2[(s3 >> 16) & 0xff] ^
				T3[(s2 >> 24)       ] ^
				k[(r2*8)-3];
			t2 =
				T0[(s2      ) & 0xff] ^
				T1[(s1 >>  8) & 0xff] ^
				T2[(s0 >> 16) & 0xff] ^
				T3[(s3 >> 24)       ] ^
				k[(r2*8)-2];
			t3 =
				T0[(s3      ) & 0xff] ^
				T1[(s2 >>  8) & 0xff] ^
				T2[(s1 >> 16) & 0xff] ^
				T3[(s0 >> 24)       ] ^
				k[(r2*8)-1];
			__LOG_TRACE__("p %d: t0 => 0x%04x",p,t0);
			__LOG_TRACE__("p %d: t1 => 0x%04x",p,t1);
			__LOG_TRACE__("p %d: t2 => 0x%04x",p,t2);
			__LOG_TRACE__("p %d: t3 => 0x%04x",p,t3);

			s0 =
				T0[(t0      ) & 0xff] ^
				T1[(t3 >>  8) & 0xff] ^
				T2[(t2 >> 16) & 0xff] ^
				T3[(t1 >> 24)       ] ^
				k[(r2*8)  ];
			s1 =
				T0[(t1      ) & 0xff] ^
				T1[(t0 >>  8) & 0xff] ^
				T2[(t3 >> 16) & 0xff] ^
				T3[(t2 >> 24)       ] ^
				k[(r2*8)+1];
			s2 =
				T0[(t2      ) & 0xff] ^
				T1[(t1 >>  8) & 0xff] ^
				T2[(t0 >> 16) & 0xff] ^
				T3[(t3 >> 24)       ] ^
				k[(r2*8)+2];
			s3 =
				T0[(t3      ) & 0xff] ^
				T1[(t2 >>  8) & 0xff] ^
				T2[(t1 >> 16) & 0xff] ^
				T3[(t0 >> 24)       ] ^
				k[(r2*8)+3];
			__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0);
			__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1);
			__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2);
			__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3);
		}

		if(key_bits >= 192) {
			key_index_sum = 8;

			t0 =
				T0[(s0      ) & 0xff] ^
				T1[(s3 >>  8) & 0xff] ^
				T2[(s2 >> 16) & 0xff] ^
				T3[(s1 >> 24)       ] ^
				k[36];
			t1 =
				T0[(s1      ) & 0xff] ^
				T1[(s0 >>  8) & 0xff] ^
				T2[(s3 >> 16) & 0xff] ^
				T3[(s2 >> 24)       ] ^
				k[37];
			t2 =
				T0[(s2      ) & 0xff] ^
				T1[(s1 >>  8) & 0xff] ^
				T2[(s0 >> 16) & 0xff] ^
				T3[(s3 >> 24)       ] ^
				k[38];
			t3 =
				T0[(s3      ) & 0xff] ^
				T1[(s2 >>  8) & 0xff] ^
				T2[(s1 >> 16) & 0xff] ^
				T3[(s0 >> 24)       ] ^
				k[39];
			__LOG_TRACE__("p %d: t0 => 0x%04x",p,t0);
			__LOG_TRACE__("p %d: t1 => 0x%04x",p,t1);
			__LOG_TRACE__("p %d: t2 => 0x%04x",p,t2);
			__LOG_TRACE__("p %d: t3 => 0x%04x",p,t3);

			s0 =
				T0[(t0      ) & 0xff] ^
				T1[(t3 >>  8) & 0xff] ^
				T2[(t2 >> 16) & 0xff] ^
				T3[(t1 >> 24)       ] ^
				k[40];
			s1 =
				T0[(t1      ) & 0xff] ^
				T1[(t0 >>  8) & 0xff] ^
				T2[(t3 >> 16) & 0xff] ^
				T3[(t2 >> 24)       ] ^
				k[41];
			s2 =
				T0[(t2      ) & 0xff] ^
				T1[(t1 >>  8) & 0xff] ^
				T2[(t0 >> 16) & 0xff] ^
				T3[(t3 >> 24)       ] ^
				k[42];
			s3 =
				T0[(t3      ) & 0xff] ^
				T1[(t2 >>  8) & 0xff] ^
				T2[(t1 >> 16) & 0xff] ^
				T3[(t0 >> 24)       ] ^
				k[43];
			__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0);
			__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1);
			__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2);
			__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3);


			if(key_bits >= 256) {
				key_index_sum = 16;

				t0 =
					T0[(s0      ) & 0xff] ^
					T1[(s3 >>  8) & 0xff] ^
					T2[(s2 >> 16) & 0xff] ^
					T3[(s1 >> 24)       ] ^
					k[44];
				t1 =
					T0[(s1      ) & 0xff] ^
					T1[(s0 >>  8) & 0xff] ^
					T2[(s3 >> 16) & 0xff] ^
					T3[(s2 >> 24)       ] ^
					k[45];
				t2 =
					T0[(s2      ) & 0xff] ^
					T1[(s1 >>  8) & 0xff] ^
					T2[(s0 >> 16) & 0xff] ^
					T3[(s3 >> 24)       ] ^
					k[46];
				t3 =
					T0[(s3      ) & 0xff] ^
					T1[(s2 >>  8) & 0xff] ^
					T2[(s1 >> 16) & 0xff] ^
					T3[(s0 >> 24)       ] ^
					k[47];
				__LOG_TRACE__("p %d: t0 => 0x%04x",p,t0);
				__LOG_TRACE__("p %d: t1 => 0x%04x",p,t1);
				__LOG_TRACE__("p %d: t2 => 0x%04x",p,t2);
				__LOG_TRACE__("p %d: t3 => 0x%04x",p,t3);

				s0 =
					T0[(t0      ) & 0xff] ^
					T1[(t3 >>  8) & 0xff] ^
					T2[(t2 >> 16) & 0xff] ^
					T3[(t1 >> 24)       ] ^
					k[48];
				s1 =
					T0[(t1      ) & 0xff] ^
					T1[(t0 >>  8) & 0xff] ^
					T2[(t3 >> 16) & 0xff] ^
					T3[(t2 >> 24)       ] ^
					k[49];
				s2 =
					T0[(t2      ) & 0xff] ^
					T1[(t1 >>  8) & 0xff] ^
					T2[(t0 >> 16) & 0xff] ^
					T3[(t3 >> 24)       ] ^
					k[50];
				s3 =
					T0[(t3      ) & 0xff] ^
					T1[(t2 >>  8) & 0xff] ^
					T2[(t1 >> 16) & 0xff] ^
					T3[(t0 >> 24)       ] ^
					k[51];
				__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0);
				__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1);
				__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2);
				__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3);
			}
		}

		t0 =
			T0[(s0      ) & 0xff] ^
			T1[(s3 >>  8) & 0xff] ^
			T2[(s2 >> 16) & 0xff] ^
			T3[(s1 >> 24)       ] ^
			k[36+key_index_sum];
		t1 =
			T0[(s1      ) & 0xff] ^
			T1[(s0 >>  8) & 0xff] ^
			T2[(s3 >> 16) & 0xff] ^
			T3[(s2 >> 24)       ] ^
			k[37+key_index_sum];
		t2 =
			T0[(s2      ) & 0xff] ^
			T1[(s1 >>  8) & 0xff] ^
			T2[(s0 >> 16) & 0xff] ^
			T3[(s3 >> 24)       ] ^
			k[38+key_index_sum];
		t3 =
			T0[(s3      ) & 0xff] ^
			T1[(s2 >>  8) & 0xff] ^
			T2[(s1 >> 16) & 0xff] ^
			T3[(s0 >> 24)       ] ^
			k[39+key_index_sum];
		__LOG_TRACE__("p %d: t0 => 0x%04x",p,t0);
		__LOG_TRACE__("p %d: t1 => 0x%04x",p,t1);
		__LOG_TRACE__("p %d: t2 => 0x%04x",p,t2);
		__LOG_TRACE__("p %d: t3 => 0x%04x",p,t3);

		// last round - save result
		s0 =
			((uint32_t)T4[(t0      ) & 0xff]      ) ^
			((uint32_t)T4[(t3 >>  8) & 0xff] <<  8) ^
			((uint32_t)T4[(t2 >> 16) & 0xff] << 16) ^
			((uint32_t)T4[(t1 >> 24)       ] << 24) ^
			k[40+key_index_sum];
		__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0);
		s1 =
			((uint32_t)T4[(t1      ) & 0xff]      ) ^
			((uint32_t)T4[(t0 >>  8) & 0xff] <<  8) ^
			((uint32_t)T4[(t3 >> 16) & 0xff] << 16) ^
			((uint32_t)T4[(t2 >> 24)       ] << 24) ^
			k[41+key_index_sum];
		__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1);
		s2 =
			((uint32_t)T4[(t2      ) & 0xff]      ) ^
			((uint32_t)T4[(t1 >>  8) & 0xff] <<  8) ^
			((uint32_t)T4[(t0 >> 16) & 0xff] << 16) ^
			((uint32_t)T4[(t3 >> 24)       ] << 24) ^
			k[42+key_index_sum];
		__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2);
		s3 =
			((uint32_t)T4[(t3      ) & 0xff]      ) ^
			((uint32_t)T4[(t2 >>  8) & 0xff] <<  8) ^
			((uint32_t)T4[(t1 >> 16) & 0xff] << 16) ^
			((uint32_t)T4[(t0 >> 24)       ] << 24) ^
			k[43+key_index_sum];
		__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3);

		/* if(m == paracrypt::BlockCipher::ECB) {
			// nothing in purpose
		}
		else */ if(m == paracrypt::BlockCipher::CBC) {
			uint32_t c0,c1,c2,c3;
			if(bi == 0) {
                // there is no previous block - use input vector
				c0 = iv[0];
				c1 = iv[1];
				c2 = iv[2];
				c3 = iv[3];
				__LOG_TRACE__("p %d (bi %d): using IV.",p,bi);
			} else {
				// previous block
				if(threadIdx.x == 0) { // && d == out) { TODO support for out-of-place
					// previous cipher-block is in another
					//  thread-block so we cannot __syncthreads()
					// and we use this data to ensure the data
					// we access is not overwritten
					int np = (blockIdx.x*4)-4;
					c0 = neigh[np  ];
					c1 = neigh[np+1];
					c2 = neigh[np+2];
					c3 = neigh[np+3];
					__LOG_TRACE__("p %d (bi %d): accesing neighbor at np %d.",p,bi,np);
				}
				else {
					c0 = d[p-4];
					c1 = d[p-3];
					c2 = d[p-2];
					c3 = d[p-1];
					__LOG_TRACE__("p %d (bi %d): accessing prev. block.",p,bi);
				}
			}
			__LOG_TRACE__("p %d: s0 => 0x%04x",p,s0);
			__LOG_TRACE__("p %d: s1 => 0x%04x",p,s1);
			__LOG_TRACE__("p %d: s2 => 0x%04x",p,s2);
			__LOG_TRACE__("p %d: s3 => 0x%04x",p,s3);
			__LOG_TRACE__("p %d: c0 => 0x%04x",p,c0);
			__LOG_TRACE__("p %d: c1 => 0x%04x",p,c1);
			__LOG_TRACE__("p %d: c2 => 0x%04x",p,c2);
			__LOG_TRACE__("p %d: c3 => 0x%04x",p,c3);
			s0 ^= c0;
			s1 ^= c1;
			s2 ^= c2;
			s3 ^= c3;
		}
		else if(m == paracrypt::BlockCipher::CFB) {
			// TODO
		}
		else if(m == paracrypt::BlockCipher::CFB || m == paracrypt::BlockCipher::GCM) {
			s0 ^= d[p  ];
			s1 ^= d[p+1];
			s2 ^= d[p+2];
			s3 ^= d[p+3];
		}
	}

	// - Do not override saving results data before others
	//  have finished working with this this block !!
	// Note: __syncthreads() has to be placed outside conditional code ...
	if(m == paracrypt::BlockCipher::CBC || m == paracrypt::BlockCipher::CFB) {
		// ... except if all threads evaluate the same condition. In this
		//  case all threads have the same mode m.
		// if(d == out) //TODO support for out of place
			__syncthreads();
	}

	if(bi < n) {
		// save result
		out[p] = s0;
		out[p+1] = s1;
		out[p+2] = s2;
		out[p+3] = s3;
	}
}

void cuda_ecb_aes_16b_encrypt(
			  paracrypt::BlockCipher::Mode m,
	  	  	  int gridSize,
	  	  	  int threadsPerBlock,
	  	  	  hipStream_t stream,
	  	  	  unsigned int n_blocks,
	  	  	  unsigned int offset,
	  	  	  unsigned char* in,
	  	  	  unsigned char* out,
	  	  	  unsigned char* neigh,
	  	  	  unsigned char* iv,
	  	  	  uint32_t* expanded_key,
	  	  	  int key_bits,
	  	  	  uint32_t* deviceTe0,
	  	  	  uint32_t* deviceTe1,
	  	  	  uint32_t* deviceTe2,
	  	  	  uint32_t* deviceTe3
	      )
{
	__cuda_ecb_aes_16b_encrypt__<<<gridSize,threadsPerBlock,0,stream>>>(
			m,
			n_blocks,
			offset,
			(const uint32_t*)in,
			(uint32_t*)out,
			(uint32_t*)neigh,
			(uint32_t*)iv,
			expanded_key,
			key_bits,
	   		deviceTe0,
	   		deviceTe1,
	   		deviceTe2,
	   		deviceTe3
	);
}

void cuda_ecb_aes_16b_decrypt(
			  paracrypt::BlockCipher::Mode m,
	  	  	  int gridSize,
	  	  	  int threadsPerBlock,
	  	  	  hipStream_t stream,
	  	  	  unsigned int n_blocks,
	  	  	  unsigned int offset,
	  	  	  unsigned char* in,
	  	  	  unsigned char* out,
	  	  	  unsigned char* neigh,
	  	  	  unsigned char* iv,
	  	  	  uint32_t* expanded_key,
	  	  	  int key_bits,
	  	  	  uint32_t* deviceTd0,
	  	  	  uint32_t* deviceTd1,
	  	  	  uint32_t* deviceTd2,
	  	  	  uint32_t* deviceTd3,
	  	  	  uint8_t* deviceTd4
	      )
{
	__cuda_ecb_aes_16b_decrypt__<<<gridSize,threadsPerBlock,0,stream>>>(
			m,
			n_blocks,
			offset,
			(const uint32_t*)in,
			(uint32_t*)out,
			(uint32_t*)neigh,
			(uint32_t*)iv,
			expanded_key,
			key_bits,
	   		deviceTd0,
	   		deviceTd1,
	   		deviceTd2,
	   		deviceTd3,
	   		deviceTd4
	);
}
